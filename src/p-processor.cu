#include "hip/hip_runtime.h"

#include <p-processor.hpp>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <InputLoopThrough.h>
#include <hip/hip_runtime.h>
#include <YUVUChroma.cuh>
#include <stdio.h>


/**** Utils *****/
inline __device__ __host__ int iDivUp( int a, int b )  		{ return (a % b != 0) ? (a / b + 1) : (a / b); }

/***************/
void PipelineObj::checkCudaError(std::string action, std::string loc)
{
	if(this->cudaStatus != hipSuccess)
	{
		std::cout<<"[Error]: Failed to "<< action<<" to"<< loc <<" \n"
				<<"[Error]: "<<hipGetErrorString(this->cudaStatus)<<std::endl;
	}
}

bool PipelineObj::toCuda(void* src, void* dst, long int size)
{
	this->cudaStatus = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
	return this->cudaStatus != hipSuccess;
}


void Processor::cudaInit()
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&this->yPackedCudaFill, this->frameSizePacked);
	if(cudaStatus != hipSuccess){
		this->checkCudaError("Allocate memory", " yPackedCudaFill");
		exit(-1);
	}

	cudaStatus = hipMalloc((void**)&this->yPackedCudaKey, this->frameSizePacked);
	if(cudaStatus != hipSuccess){
		this->checkCudaError("Allocate memory", " yPackedCudaKey");
		exit(-1);
	}

	cudaStatus = hipMalloc((void**)&this->yPackedCudaVideo, this->frameSizePacked);
	if(cudaStatus != hipSuccess){
		this->checkCudaError("Allocate memory", " yPackedCudaVideo");
		exit(-1);
	}

	cudaStatus = hipMalloc((void**)&this->yUnpackedCudaFill, this->frameSizeUnpacked);
	if(cudaStatus != hipSuccess){
		this->checkCudaError("Allocate memory", " yUnpackedCudaFill");
		exit(-1);
	}

	cudaStatus = hipMalloc((void**)&this->yUnpackedCudaKey, this->frameSizeUnpacked);
	if(cudaStatus != hipSuccess){
		this->checkCudaError("Allocate memory", " yUnpackedCudaKey");
		exit(-1);
	}

	cudaStatus = hipMalloc((void**)&this->yUnpackedCudaVideo, this->frameSizeUnpacked);
	if(cudaStatus != hipSuccess){
		this->checkCudaError("Allocate memory", " yUnpackedCudaVideo");
		exit(-1);
	}

	cudaStatus = hipMalloc((void**)&this->cudaRGB, this->iWidth*this->iHeight*sizeof(uchar3));
	if(cudaStatus != hipSuccess){
		this->checkCudaError("Allocate memory", " cudaRGB");
		exit(-1);
	}

	std::cout<<"[Info]: Finished initializing cuda variables\n"<<std::endl;
}


void Processor::sendDataTo()
{
	// read video from the deck Link card and send it to cuda
	// retrive frame if there's one to retrieve
	this->deckLinkInput->WaitForFrames(this->iDelayFrames);
	bool popVid = this->iDelayFrames <= this->deckLinkInput->imagelistVideo.GetFrameCount();
	static void* videoFrame;

	if(videoFrame)
		free(videoFrame);

	videoFrame = this->deckLinkInput->imagelistVideo.GetFrame(true);
	void* keyFrame = this->deckLinkInput->imagelistKey.GetFrame(true);
	void* fillFrame = this->deckLinkInput->imagelistFill.GetFrame(true);

	hipError_t cudaStatus;

	if(videoFrame && keyFrame && fillFrame)
	{
		if(this->toCuda((void*)videoFrame,(void*)this->yPackedCudaVideo, this->frameSizePacked))
		{
			this->checkCudaError("Copy data", " yPackedCudaVideo");
			exit(-1);
		}

		if(this->toCuda(keyFrame, this->yPackedCudaKey, this->frameSizePacked))
		{
			this->checkCudaError("Copy data", " yPackedCudaKey");
			exit(-1);
		}

		if(this->toCuda(fillFrame, this->yPackedCudaFill, this->frameSizePacked))
		{
			this->checkCudaError("Copy data", " yPackedCudaFill");
			exit(-1);
		}
	}

	if(fillFrame)
		free(fillFrame);

	if(keyFrame)
		free(keyFrame);

//	std::cout<<"[info]: Done sending data to cuda"<<std::endl;
}


void Processor::unpackYUV()
{
	const dim3 block(16, 16);
	const dim3 grid(iDivUp(this->deckLinkInput->m_RowLength/SIZE_ULONG4_CUDA, block.x), iDivUp(this->iHeight, block.y));
	const int srcAlignedWidth = this->deckLinkInput->m_RowLength/SIZE_ULONG4_CUDA;
	const int dstAlignedWidth = this->iWidth/2;

	// Unpack yuv video from decklink and store it in yUnpackedCudaVideo
	yuyvPackedToyuyvUnpacked <<<grid, block, 0, this->stream>>>(
			(uint4*)this->yPackedCudaVideo,
			(uint4*)this->yUnpackedCudaVideo,
			srcAlignedWidth,
			dstAlignedWidth,
			this->iHeight
		);

	this->cudaStatus = hipDeviceSynchronize();
	this->checkCudaError("synchronize device", " at unpacking");
	// Unpack yuv key from decklink and store it in yUnpackedCudaKey
	yuyvPackedToyuyvUnpacked <<<grid, block, 0, this->stream>>>(
				(uint4*)this->yPackedCudaKey,
				(uint4*)this->yUnpackedCudaKey,
				srcAlignedWidth,
				dstAlignedWidth,
				this->iHeight
			);
	this->cudaStatus = hipDeviceSynchronize();
	this->checkCudaError("synchronize device", " at unpacking");

	// Unpack yuv fill from decklink and store it in yUnpackedCudaFill
	yuyvPackedToyuyvUnpacked <<<grid, block, 0, this->stream>>>(
				(uint4*)this->yPackedCudaFill,
				(uint4*)this->yUnpackedCudaFill,
				srcAlignedWidth,
				dstAlignedWidth,
				this->iHeight
			);

	this->cudaStatus = hipDeviceSynchronize();
	this->checkCudaError("synchronize device", " at unpacking");
}

void Processor::snapshot(cv::cuda::GpuMat* RGBData)
{
	if(this->mtx)
		this->mtx->lock();

	this->cudaReset();
	const int srcAlignedWidth = this->deckLinkInput->m_RowLength/SIZE_ULONG4_CUDA;
	const int dstAlignedWidth = this->iWidth/2;
	const dim3 block(16, 16);
	const dim3 grid(iDivUp(dstAlignedWidth, block.x), iDivUp(this->iHeight, block.y));
//

	uint4* video2;
	hipMalloc(&video2, this->frameSizeUnpacked);

	yuyvUmPackedToRGB_lookup <<<grid, block , 0, this->stream>>> (
			(uint4*)this->yUnpackedCudaVideo,
			this->cudaRGB,
			dstAlignedWidth,
			this->iWidth,
			this->iHeight,
//			(uint4*)this->yUnpackedCudaKey,
			video2,
			nullptr // this variable is not used in the function
		);
//
	this->cudaStatus = hipDeviceSynchronize();
	this->checkCudaError("synchronize device", " at yUyVUnpackedToRGB");

	this->cudaStatus = hipMemcpy(RGBData->data, (uchar*)this->cudaRGB, this->iWidth*this->iHeight*sizeof(uchar3), hipMemcpyDeviceToDevice);
	this->checkCudaError("copy memory", " from cudaRGB to RGBData->data");

//	std::cout<<"[info]: Finished copying snapshot\n";
	hipFree(video2);

	if(this->mtx)
			this->mtx->unlock();
}

void Processor::cudaReset()
{
	this->cudaStatus = hipMemset(this->cudaRGB, 0, this->iHeight*this->iWidth*sizeof(uchar));
	this->checkCudaError("reset", "cudaRGB");
}

void Processor::cudaCleanup()
{
	this->cudaStatus = hipFree(this->yPackedCudaFill);
	this->checkCudaError("free", "yPackedCudaFill");

	this->cudaStatus = hipFree(this->yPackedCudaVideo);
	this->checkCudaError("free", "yPackedCudaVideo");

	this->cudaStatus = hipFree(this->yPackedCudaKey);
	this->checkCudaError("free", "yUnpackedCudaKey");

	this->cudaStatus = hipFree(this->yUnpackedCudaFill);
	this->checkCudaError("free", "yUnpackedCudaFill");

	this->cudaStatus = hipFree(this->yUnpackedCudaVideo);
	this->checkCudaError("free", "yUnpackedCudaVideo");

	this->cudaStatus = hipFree(this->yUnpackedCudaKey);
	this->checkCudaError("free", "yUnpackadCudaKey");

	this->cudaStatus = hipFree(this->cudaRGB);
	this->checkCudaError("free", "cudaRGB");

}

void Processor::run()
{
	if(this->mtx)
		this->mtx->lock();

	this->sendDataTo();
	this->unpackYUV();
	if(this->mtx)
		this->mtx->unlock();
}

void ChrommaKey::cudaInit()
{
	int i = 0;
	this->cudaStatus = hipMalloc(this->chromaGeneratedMask, 3*sizeof(uchar*));
	this->checkCudaError("Allocate memory", "chromeGeneratedMask");

	for(i=0; i<3; i++)
	{
		this->cudaStatus = hipMalloc(this->chromaGeneratedMask, this->iHeight*this->iWidth*sizeof(uchar));
		this->checkCudaError("Allocate memory", "chromeGeneratedMask");
	}
}
