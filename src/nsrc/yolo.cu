#include "hip/hip_runtime.h"
/*
 * yolo.cpp
 *
 *  Created on: 12 Jun 2023
 *      Author: jurie
 */

#include "yolo.hpp"
#include <fstream>

#include "config.h"
#include "cuda_utils.h"
#include "logging.h"
#include "utils.h"
#include "preprocess.h"
#include "postprocess.h"
#include "model.h"

using namespace nvinfer1;

static Logger gLogger;





YoloMask::YoloMask(IPipeline *obj): IMask(obj) // @suppress("Class members should be properly initialized")
{
	char *cwd = getenv("CWD");
	std::string rootDir(cwd);
	std::string engine_name = rootDir+"/res/yolo-seg-4.engine";

	api = new YoloAPI(obj, std::string("/home/jurie/Documents/Computer Vision/tensorrtx/yolov5/build/yolov5x-latest.engine"));

	this->outputBufferDetections = nullptr;
	this->outputBufferMask = nullptr;
	this->batchData = nullptr;
	this->context = nullptr;
	this->engine = nullptr;
	this->runtime = nullptr;
	this->stream;
	this->cudaStatus = hipStreamCreate(&stream);
	this->checkCudaError("create cuda stream", "Yolo Mask Constructor");
	this->started = false;
	this->loaded = false;
	this->maskOutCpu = new float[kBatchSize * kOutputSize2];
	this->detectionsOutCpu = new float[kBatchSize * kOutputSize1];
	memset(this->detectionsOutCpu, 0, kBatchSize * kOutputSize1*sizeof(float));
	memset(this->maskOutCpu, 0 , kBatchSize * kOutputSize2*sizeof(float));
	this->initialize();
}

void YoloMask::initialize()
{
//	this->started = false;
//	hipSetDevice(kGpuId);

//

//	std::ifstream engine_file(engine_name, std::ios::binary);
//
//	if(!engine_file)
//	{
//		std::cerr<<"Engine File doesn't exist: \n"<<"Path: "<<engine_name<<std::endl;
//		return;
//	}
//
//	deserialize_engine(engine_name, &runtime, &engine, &context);
//
//	assert(engine->getNbBindings() == 3);
//	// In order to bind the buffers, we need to know the names of the input and output tensors.
//	// Note that indices are guaranteed to be less than IEngine::getNbBindings()
//	const int inputIndex = engine->getBindingIndex(kInputTensorName);
//	const int outputIndex1 = engine->getBindingIndex(kOutputTensorName);
//	const int outputIndex2 = engine->getBindingIndex("proto"); // mask
//	assert(inputIndex == 0);
//	assert(outputIndex1 == 1);
//	assert(outputIndex2 == 2);

//	initYolo();

	this->started = true;
}

void YoloMask::__cutToPanels()
{
	// input is fixed to width*2 and height/2 (Interlacing problem)
	int n = 3;
	int width = this->frame.cols/(2*n);
	int overlappingFactor = width/n+ width%n;
	int lastEnd = 0;

	for(int i=0; i<(n+1); i++)
	{
		cv::Rect roi(cv::Point(lastEnd, 0), cv::Size(width, this->frame.rows));
		cv::Mat tImg = this->frame(roi);
		cv::resize(tImg, tImg, cv::Size(640, 640));
		this->img_batch.push_back(tImg);
		lastEnd += width-overlappingFactor;
	}

}

void YoloMask::prepareImages()
{
	// convert to RGB,
	// cut to 4 panels
	// send it to yolo
	static int tracker = 0;
	std::cout<<tracker<<std::endl;
	this->img_batch.clear();
	this->convertToRGB();
	cv::cuda::GpuMat mat(cv::Size(this->iWidth, this->iHeight), CV_8UC3, this->rgbVideo);

	mat.download(this->frame);
	this->frame.create(cv::Size(this->iWidth*2, this->iHeight/2), CV_8UC3);
	this->__cutToPanels();


}

void YoloMask::preprocess()
{
	int src_height  = this->iHeight;
	int src_width = this->iWidth;

	int dst_width = kInputW;
	int dst_height = kInputH;

	AffineMatrix s2d, d2s;
	float scale = std::min(kInputH / (float)src_height, kInputW / (float)src_width);

	s2d.value[0] = scale;
	s2d.value[1] = 0;
	s2d.value[2] = -scale * src_width  * 0.5  + dst_width * 0.5;
	s2d.value[3] = 0;
	s2d.value[4] = scale;
	s2d.value[5] = -scale * src_height * 0.5 + dst_height * 0.5;

	cv::Mat m2x3_s2d(2, 3, CV_32F, s2d.value);
	cv::Mat m2x3_d2s(2, 3, CV_32F, d2s.value);
	cv::invertAffineTransform(m2x3_s2d, m2x3_d2s);

	memcpy(d2s.value, m2x3_d2s.ptr<float>(0), sizeof(d2s.value));

	int jobs = kInputH * kInputW;
	int threads = 256;
	int blocks = ceil(jobs / (float)threads);

	cv::cuda::GpuMat rgbData(this->iHeight, this->iWidth, CV_8UC3, this->rgbVideo, this->iWidth*sizeof(uchar3));

	warpaffine_kernel<<<blocks, threads, 0, stream>>>(
	  rgbData.ptr(), src_width * 3, src_width,
	  src_height, this->gpuBuffs[0], dst_width,
	  dst_height, 128, d2s, jobs);
}

void YoloMask::runInference()
{
//	float *buffers[3];
//	buffers[0] = this->batchData;
//	buffers[1] = this->outputBufferDetections;
//	buffers[2] = this->outputBufferMask;


	context->enqueue(kBatchSize,(void**)this->gpuBuffs, stream, nullptr);

	this->cudaStatus = hipMemcpyAsync(this->detectionsOutCpu, this->gpuBuffs[1], kBatchSize * kOutputSize1 * sizeof(float), hipMemcpyDeviceToHost, stream);
	this->checkCudaError("Copy memory", "cpu memory");
	this->cudaStatus = hipMemcpyAsync(this->maskOutCpu, this->gpuBuffs[2], kBatchSize * kOutputSize2 * sizeof(float), hipMemcpyDeviceToHost, stream);
	this->checkCudaError("Copy memory", "cpu memory");
	hipDeviceSynchronize();
	this->cudaStatus = hipGetLastError();
	this->checkCudaError("synchronize device", "cpu memory");
}

void YoloMask::postprocess()
{
	// NMS
//	std::vector<std::vector<Detection>> res_batch;
//	batch_nms(res_batch, this->detectionsOutCpu, kBatchSize, kOutputSize1, kConfThresh, kNmsThresh);
//
//	for (size_t b = 0; b < kBatchSize; b++)
//	{
////		auto& res = res_batch[b];
////		auto masks = process_mask_s(&this->maskOutCpu[b * kOutputSize2], kOutputSize2, res);
//	}


}


void YoloMask::create()
{
	if(!this->loaded) return;
	if(!this->started)return;
	this->preprocess();
	this->runInference();
	this->postprocess();
}

uchar* YoloMask::output()
{
	return this->maskBuffer;
}

bool YoloMask::isMask()
{
	return this->mask;
}
void YoloMask::load(float* bD, float* oD, float* oM)
{
	this->batchData = bD;
	this->outputBufferDetections = oD;
	this->outputBufferMask = oM;
	this->loaded = true;
}


void YoloMask::getBatch()
{
	hipSetDevice(kGpuId);
	this->prepareImages();
	api->run(this->img_batch);
}


char *cwd = getenv("CWD");
std::string rootDir(cwd);
std::string engine_name = rootDir+"/res/yolo-seg-4.engine";
  // Deserialize the engine from file

IRuntime* runtime = nullptr;
ICudaEngine* engine = nullptr;
IExecutionContext* context = nullptr;
hipStream_t stream;
// Prepare cpu and gpu buffers
float* gpu_buffers[3];
float* cpu_output_buffer1 = nullptr;
float* cpu_output_buffer2 = nullptr;


void initYolo()
{
	hipSetDevice(kGpuId);
	CUDA_CHECK(hipStreamCreate(&stream));
	deserialize_engine(engine_name, &runtime, &engine, &context);
	 // Init CUDA preprocessing
	cuda_preprocess_init(kMaxInputImageSize);
	prepare_buffers(engine, &gpu_buffers[0], &gpu_buffers[1], &gpu_buffers[2], &cpu_output_buffer1, &cpu_output_buffer2);
}


void yoloRun(std::vector<cv::Mat> img_batch) {


  int counter = 0;

  // Preprocess
  cuda_batch_preprocess(img_batch, gpu_buffers[0], kInputW, kInputH, stream);

    // Run inference
//  std::cout<<"I execute"<<std::endl;
   auto start = std::chrono::system_clock::now();
   infer(*context, stream, (void**)gpu_buffers, cpu_output_buffer1, cpu_output_buffer2, kBatchSize);
   auto end = std::chrono::system_clock::now();
   std::cout << "inference time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << "ms" << std::endl;

    // NMS
   std::vector<std::vector<Detection>> res_batch;
   batch_nms(res_batch, cpu_output_buffer1, img_batch.size(), kOutputSize1, kConfThresh, kNmsThresh);


	// Draw result and save image
   for (size_t b = 0; b < img_batch.size(); b++) {
	  auto& res = res_batch[0];
	  cv::Mat img = img_batch[b];

	  auto masks = process_mask(&cpu_output_buffer2[b * kOutputSize2], kOutputSize2, res);
//
	  draw_mask_bbox(img, res, masks);
//	  break;
//	  cv::imwrite("_" + img_name_batch[b], img);
	}


  // Release stream and buffers
//  hipStreamDestroy(stream);
//  CUDA_CHECK(hipFree(gpu_buffers[0]));
//  CUDA_CHECK(hipFree(gpu_buffers[1]));
//  CUDA_CHECK(hipFree(gpu_buffers[2]));
//  delete[] cpu_output_buffer1;
//  delete[] cpu_output_buffer2;
//  cuda_preprocess_destroy();
//  // Destroy the engine
//  context->destroy();
//  engine->destroy();
//  runtime->destroy();

}





