#include "hip/hip_runtime.h"
/*
 * pipeline.cpp
 *
 *  Created on: 31 May 2023
 *      Author: jurie
 */
//#include <p-processor.hpp>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <InputLoopThrough.h>
#include <hip/hip_runtime.h>
#include <YUVUChroma.cuh>
#include <stdio.h>
#include <opencv2/cudafilters.hpp>
#include "interfaces.hpp"

/**** Utils *****/
inline __device__ __host__ int iDivUp( int a, int b )  		{ return (a % b != 0) ? (a / b + 1) : (a / b); }


IPipeline::IPipeline()
{
	this->fill = nullptr;
	this->key = nullptr;
	this->video = nullptr;
	this->augVideo = nullptr;
	this->frameSizePacked = 0;
	this->frameSizeUnpacked = 0;
	this->iHeight = 0;
	this->iWidth = 0;
	this->cudaStatus ;
	this->cudaStatus = hipStreamCreate(&this->stream);
	this->mtx = nullptr;
}

IPipeline::IPipeline(IPipeline* pObj)
{
	assert(pObj!=nullptr);
	this->fill = pObj->fill;
	this->key = pObj->key;
	this->video = pObj->video;
	this->augVideo = pObj->augVideo;
	this->frameSizePacked = pObj->frameSizePacked;
	this->frameSizeUnpacked = pObj->frameSizeUnpacked;
	this->iHeight = pObj->iHeight;
	this->iWidth = pObj->iWidth;
	this->cudaStatus ;
	this->cudaStatus = hipStreamCreate(&this->stream);
	this->mtx = pObj->mtx;
}

void IPipeline::checkCudaError(std::string action, std::string loc)
{
	if(this->cudaStatus != hipSuccess)
	{
		std::cerr<<"[Error]: Failed to "<< action<<" to "<< loc <<" \n"
				<<"[Error]: "<<hipGetErrorString(this->cudaStatus)
		<<std::endl;
	}
}


Input::Input(VideoIn* i): IPipeline()
{
	this->input = i;
	this->in = false;
	this->pFill = nullptr;
	this->pKey = nullptr;
	this->pVideo = nullptr;
}

void Input::init()
{
	while(this->input->m_sizeOfFrame == -1)
		std::this_thread::sleep_for(std::chrono::milliseconds(40));

	this->cudaStatus = hipMalloc(&this->pVideo, this->input->m_sizeOfFrame);
	this->checkCudaError("Allocate Memory", " packedVideo");

	this->cudaStatus = hipMalloc(&this->pKey, this->input->m_sizeOfFrame);
	this->checkCudaError("Allocate Memory", " packedKey");

	this->cudaStatus = hipMalloc(&this->pFill, this->input->m_sizeOfFrame);
	this->checkCudaError("Allocate Memory", " packedVideo");

	this->iHeight = this->input->m_iHeight;
	this->iWidth = this->input->m_iWidth;
	this->frameSizePacked = this->input->m_sizeOfFrame;
	this->frameSizeUnpacked = this->input->m_iFrameSizeUnpacked;
}


void Input::run()
{
	static void* videoFrame;

	if(videoFrame)
			free(videoFrame);
	videoFrame = this->input->imagelistVideo.GetFrame(true);
	void* keyFrame = this->input->imagelistKey.GetFrame(true);
	void* fillFrame = this->input->imagelistFill.GetFrame(true);

	this->cudaStatus = hipMemcpy(this->pVideo, videoFrame, this->frameSizePacked, hipMemcpyHostToDevice);
	assert((this->cudaStatus == hipSuccess));

	this->cudaStatus = hipMemcpy(this->pKey, keyFrame, this->frameSizePacked, hipMemcpyHostToDevice);
	assert((this->cudaStatus == hipSuccess));

	this->cudaStatus = hipMemcpy(this->pFill, fillFrame, this->frameSizePacked, hipMemcpyHostToDevice);
	assert((this->cudaStatus == hipSuccess));

	if(fillFrame)
			free(fillFrame);

	if(keyFrame)
		free(keyFrame);
}




Preprocessor::Preprocessor(uchar2* video, uchar2*key, uchar2*fill) // these variables must be GPU pointers
{
	this->pVideo = video;
	this->pKey = key;
	this->pFill = fill;
	this->augVideo = nullptr;
	this->rgbVideo = nullptr;
}

Preprocessor::Preprocessor(IPipeline* obj, uchar2* video, uchar2*key, uchar2*fill): IPipeline(obj)
{
//	this->pVideo = obj->pVideo;
	this->pVideo = video;
	this->pKey = key;
	this->pFill = fill;
	this->augVideo = nullptr;
	this->rgbVideo = nullptr;
}

void Preprocessor::init()
{
	this->cudaStatus = hipMalloc((void**)&this->video, this->frameSizeUnpacked);
	this->checkCudaError("Allocate Memory", " video buffer");
	assert((this->cudaStatus == hipSuccess));

	this->cudaStatus = hipMalloc((void**)&this->key, this->frameSizeUnpacked);
	this->checkCudaError("Allocate Memory", " key buffer");
	assert((this->cudaStatus == hipSuccess));

	this->cudaStatus = hipMalloc((void**)&this->fill, this->frameSizeUnpacked);
	this->checkCudaError("Allocate Memory", " fill buffer");
	assert((this->cudaStatus == hipSuccess));

	this->cudaStatus = hipMalloc((void**)&this->augVideo, this->frameSizeUnpacked);
	this->checkCudaError("Allocate Memory", " augmented video buffer");
	assert((this->cudaStatus == hipSuccess));

	this->cudaStatus = hipMalloc((void**)&this->rgbVideo, this->iHeight*this->iWidth*sizeof(uchar3));
	this->checkCudaError("Allocate Memory", " RGB Video buffer");
	assert((this->cudaStatus == hipSuccess));
}

void Preprocessor::unpack()
{
	// Unpacked yuv to yuyv
	const dim3 block(16, 16);
	const dim3 grid(iDivUp(this->iWidth, block.x), iDivUp(this->iHeight, block.y));
	const int srcAlignedWidth = this->iWidth;
	const int dstAlignedWidth = this->iWidth/2;

	// Unpack yuv video from decklink and store it in yUnpackedCudaVideo
	yuyvPackedToyuyvUnpacked <<<grid, block>>>(
			(uint4*)this->pVideo,
			(uint4*)this->video,
			srcAlignedWidth,
			dstAlignedWidth,
			this->iHeight
		);

	// Unpack yuv key from decklink and store it in yUnpackedCudaKey
	yuyvPackedToyuyvUnpacked <<<grid, block, 0, this->stream>>>(
				(uint4*)this->pKey,
				(uint4*)this->key,
				srcAlignedWidth,
				dstAlignedWidth,
				this->iHeight
			);


	// Unpack yuv fill from decklink and store it in yUnpackedCudaFill
	yuyvPackedToyuyvUnpacked <<<grid, block>>>(
				(uint4*)this->pFill,
				(uint4*)this->fill,
				srcAlignedWidth,
				dstAlignedWidth,
				this->iHeight
			);


	this->cudaStatus = hipDeviceSynchronize();
	this->checkCudaError("synchronize device", " at unpacking");

	this->cudaStatus = hipMemcpyAsync(this->augVideo, this->video, this->frameSizeUnpacked, hipMemcpyDeviceToHost,this->stream);
	this->checkCudaError("copy data", " augmented video buffer");

}

void Preprocessor::convertToRGB()
{

}

void Preprocessor::create()
{

}

LookupTable::LookupTable(IPipeline *obj): IPipeline(obj)
{
	this->lookupBuffer = nullptr;
	this->loaded = false;
}

void LookupTable::create()
{
	this->cudaStatus = hipMalloc((void**)&this->lookupBuffer, (long int)4*pow(2, 10)*sizeof(uchar));
	assert((this->cudaStatus==hipSuccess));
	this->loaded = false;
}

void LookupTable::update(bool init, bool clickEn, MouseData md, WindowSettings ws)
{
//	std::cout<<"[info]: Update Lookup started"<<std::endl;

	if(!clickEn)return;

	if (md.bHandleLDown)
	{
		this->loaded = false;
		int maxRecSize = 200;
		float ScalingValue = maxRecSize*1.0/ws.m_iOuter_Diam*1.0;

		const dim3 block(16, 16);
		const dim3 grid(
				iDivUp((ws.m_iOuter_Diam+ws.m_iUV_Diam)*2, block.x),
				iDivUp((ws.m_iOuter_Diam+ws.m_iUV_Diam)*2, block.y)
				);

		uchar* ptrLookUpDataToUse = this->lookupBuffer;

		this->mtx->lock();
		std::cout<<"[info]: Thread locked ..."<<std::endl;

		for (int x = (md.iXUpDynamic / 2); x<(md.iXDownDynamic/2); x++)
		{
			for (int y = md.iYUpDynamic; y < md.iYDownDynamic; y=y+2)
			{
				UpdateLookupFrom_XY_Posision_Diffrent_Scaling <<<grid, block>>> (
						(uint4*)this->augVideo,
						ptrLookUpDataToUse,
						x, y,
						(this->iHeight / 2),
						ws.m_iOuter_Diam*2,
						ws.m_iUV_Diam*2,
						ws.m_iLum_Diam,
						ScalingValue,
						maxRecSize
						);
			}
		}

		this->cudaStatus = hipDeviceSynchronize();
		this->checkCudaError("synchronize host", " kernel: updateLookupFromMouse");
		assert(this->cudaStatus==hipSuccess);
		this->loaded = true;
		this->mtx->unlock();
	}
//	std::cout<<"[info]: LookupTable updated successfully"<<std::endl;
}

void IMask::init()
{
	this->cudaStatus = hipMalloc((void**)this->maskBuffer, this->iHeight*this->iWidth*sizeof(uchar));
	assert(this->cudaStatus==hipSuccess);
}

void IMask::erode(int size)
{
	cv::cuda::GpuMat chrommaMaskInput;
	cv::cuda::GpuMat chrommaMaskOutput(this->iWidth/2,this->iHeight*2,CV_8UC1, this->maskBuffer,Mat::CONTINUOUS_FLAG);
	chrommaMaskOutput.step=this->iWidth*2;

	// erode output mask
	int an = size;
	cv::Mat element = getStructuringElement(MORPH_ELLIPSE, Size(an*2+1, an*2+1), Point(an, an));
	Ptr<cv::cuda::Filter> erodeFilter = cv::cuda::createMorphologyFilter(MORPH_ERODE, chrommaMaskInput.type(), element);
	erodeFilter->apply(chrommaMaskInput, chrommaMaskOutput);
}

void IMask::dilate(int size)
{
	cv::cuda::GpuMat chrommaMaskInput;
	cv::cuda::GpuMat chrommaMaskOutput(this->iWidth/2,this->iHeight*2,CV_8UC1, this->maskBuffer,Mat::CONTINUOUS_FLAG);
	chrommaMaskOutput.step=this->iWidth*2;

	// Dilate the output mask
	int an = size;
	cv::Mat element = getStructuringElement(MORPH_ELLIPSE, Size(an*2+1, an*2+1), Point(an, an));
	Ptr<cv::cuda::Filter> erodeFilter2 = cv::cuda::createMorphologyFilter(MORPH_DILATE, chrommaMaskInput.type(), element);
	erodeFilter2->apply(chrommaMaskInput, chrommaMaskOutput);
}

ChrommaMask::ChrommaMask(IPipeline* obj, LookupTable* t): IMask(obj)
{
	this->table = t;
}

void ChrommaMask::create()
{
	if(!table->isLoaded())return;
	this->mtx->lock();
	this->mask = false;
	const int dstAlignedWidth = this->iWidth;
	const int srcAlignedWidth = this->iWidth/2;
	const dim3 block(16, 16);
	const dim3 grid(iDivUp(srcAlignedWidth, block.x), iDivUp(this->iHeight, block.y));

	yuyv_Unpacked_GenerateMask <<<grid, block, 0, this->stream>>> (
			(uint4*)this->augVideo,
			this->maskBuffer,
			this->table->output(),
			this->iWidth,
			this->iHeight,
			srcAlignedWidth,
			dstAlignedWidth,
			0
			);

	this->cudaStatus = hipDeviceSynchronize();
	this->checkCudaError("synchronize host", "yuyvGenerateMask");
	assert((this->cudaStatus == hipSuccess));

	this->mask = true;

	this->mtx->unlock();
}

uchar* ChrommaMask::output()
{
	if(!this->mask)return nullptr;
	this->create();
	this->update();
	return this->maskBuffer;
}

void YoloMask::create()
{

}

uchar* YoloMask::output()
{
	return this->maskBuffer;
}

void Mask::create()
{
	// perform some mask creation here ...
}

uchar* Mask::output()
{
	return this->chromma->maskBuffer;
}




