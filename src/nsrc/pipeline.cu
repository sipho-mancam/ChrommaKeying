#include "hip/hip_runtime.h"
/*
 * pipeline.cpp
 *
 *  Created on: 31 May 2023
 *      Author: jurie
 */
//#include <p-processor.hpp>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <InputLoopThrough.h>
#include <hip/hip_runtime.h>
#include <YUVUChroma.cuh>
#include <stdio.h>
#include <opencv2/cudafilters.hpp>
#include "interfaces.hpp"
#include <ui.hpp>

/**** Utils *****/
inline __device__ __host__ int iDivUp( int a, int b )  		{ return (a % b != 0) ? (a / b + 1) : (a / b); }


IPipeline::IPipeline()
{
	this->fill = nullptr;
	this->key = nullptr;
	this->video = nullptr;
	this->augVideo = nullptr;
	this->rgbVideo = nullptr;
	this->frameSizePacked = 0;
	this->frameSizeUnpacked = 0;
	this->rowLength = 0;
	this->iHeight = 0;
	this->iWidth = 0;
	this->cudaStatus ;
	this->cudaStatus = hipStreamCreate(&this->stream);
	this->mtx = nullptr;
}

IPipeline::IPipeline(IPipeline* pObj)
{
	assert(pObj!=nullptr);
	this->fill = pObj->fill;
	this->key = pObj->key;
	this->video = pObj->video;
	this->augVideo = pObj->augVideo;
	this->rgbVideo = pObj->rgbVideo;
	this->frameSizePacked = pObj->frameSizePacked;
	this->frameSizeUnpacked = pObj->frameSizeUnpacked;
	this->iHeight = pObj->iHeight;
	this->iWidth = pObj->iWidth;
	this->cudaStatus ;
	this->cudaStatus = hipStreamCreate(&this->stream);
	this->mtx = pObj->mtx;
	this->rowLength  = pObj->rowLength;
}

void IPipeline::checkCudaError(std::string action, std::string loc)
{
	if(this->cudaStatus != hipSuccess)
	{
		std::cerr<<"[Error]: Failed to "<< action<<" to "<< loc <<" \n"
				<<"[Error]: "<<hipGetErrorString(this->cudaStatus)
		<<std::endl;
	}
}

void IPipeline::convertToRGB()
{

	const dim3 block(16, 16);
	const dim3 grid(iDivUp(this->iWidth/2, block.x), iDivUp(this->iHeight, block.y));


	yuyvUnpackedToRGB<<<grid, block>>>(
			this->augVideo,
			this->rgbVideo,
			this->iWidth/2,
			this->iWidth,
			this->iHeight,
			this->key
		);

	this->cudaStatus = hipGetLastError();
	this->checkCudaError("Launch Kernel", "Device");

	this->cudaStatus = hipDeviceSynchronize();
	this->checkCudaError("Synchronize device", " host");

}


Input::Input(VideoIn* i): IPipeline()
{
	this->input = i;
	this->in = false;
	this->pFill = nullptr;
	this->pKey = nullptr;
	this->pVideo = nullptr;
	this->init();
}

void Input::init()
{
	while(this->input->m_sizeOfFrame == -1)
		std::this_thread::sleep_for(std::chrono::milliseconds(40));

//	std::this_thread::sleep_for(std::chrono::milliseconds(100));
	input->WaitForFrames(-1);

	input->imagelistVideo.ClearAll(0);
	input->imagelistFill.ClearAll(0);
	input->imagelistKey.ClearAll(0);
	input->ImagelistOutput.ClearAll(1);

	this->iHeight = this->input->m_iHeight;
	this->iWidth = this->input->m_iWidth;
	this->frameSizePacked = this->input->m_sizeOfFrame;
	this->frameSizeUnpacked = this->input->m_iFrameSizeUnpacked;
	this->rowLength = this->input->m_RowLength;
}

void Input::load(uchar2* pv, uchar2* pk, uchar2* pf)
{
	this->pVideo = pv; this->pKey = pk; this->pFill = pf;
}


void Input::run()
{
	const dim3 block(16, 16);
	const dim3 grid(iDivUp(this->rowLength/16, block.x), iDivUp(this->iHeight, block.y));
	const int srcAlignedWidth = this->rowLength/16;
	const int dstAlignedWidth = this->iWidth/2;

	input->WaitForFrames(1);
	static void* videoFrame;
	this->in = false;
	if(videoFrame)
		free(videoFrame);
	videoFrame = this->input->imagelistVideo.GetFrame(true);
	void* keyFrame = this->input->imagelistKey.GetFrame(true);
	void* fillFrame = this->input->imagelistFill.GetFrame(true);

	this->cudaStatus = hipMemcpy(this->pVideo, videoFrame, this->frameSizePacked, hipMemcpyHostToDevice);
	this->checkCudaError("copy memory", " pVideo");
//	std::cout<<"Video: "<<videoFrame<<std::endl;
	assert((this->cudaStatus == hipSuccess));

	this->cudaStatus = hipMemcpy(this->pKey, keyFrame, this->frameSizePacked, hipMemcpyHostToDevice);
	this->checkCudaError("copy memory", " pKey");
	assert((this->cudaStatus == hipSuccess));

	this->cudaStatus = hipMemcpy(this->pFill, fillFrame, this->frameSizePacked, hipMemcpyHostToDevice);
	this->checkCudaError("copy memory", " pFill");
	assert((this->cudaStatus == hipSuccess));


	if(fillFrame)
		free(fillFrame);
	if(keyFrame)
		free(keyFrame);
	this->in = true;
}


Preprocessor::Preprocessor(uchar2* video, uchar2*key, uchar2*fill) // these variables must be GPU pointers
{
	this->pVideo = video;
	this->pKey = key;
	this->pFill = fill;
	this->augVideo = nullptr;
	this->rgbVideo = nullptr;
	this->init();
}

Preprocessor::Preprocessor(IPipeline* obj, uchar2* video, uchar2*key, uchar2*fill): IPipeline(obj)
{
	this->pVideo = video;
	this->pKey = key;
	this->pFill = fill;
	this->augVideo = nullptr;
	this->rgbVideo = nullptr;
	this->init();
}

void Preprocessor::init()
{

}

void Preprocessor::reload(uchar2* pv, uchar2* pk, uchar2* pf)
{
	this->pVideo = pv; this->pKey = pk; this->pFill = pf;
}

void Preprocessor::load(uint4* v, uint4* k, uint4* f, uint4* av, uchar3* rgb)
{
	this->video = v; this->key = k; this->fill = f; this->augVideo = av; this->rgbVideo = rgb;
}

void Preprocessor::unpack()
{
	// Unpacked yuv to yuyv
	const dim3 block(16, 16);
	const dim3 grid(iDivUp(this->rowLength/16, block.x), iDivUp(this->iHeight, block.y));
	const int srcAlignedWidth = this->rowLength/16;
	const int dstAlignedWidth = this->iWidth/2;

	// Unpack yuv video from decklink and store it in yUnpackedCudaVideo
	yuyvPackedToyuyvUnpacked <<<grid, block>>>(
			(uint4*)this->pVideo,
			this->video,
			srcAlignedWidth,
			dstAlignedWidth,
			this->iHeight
		);
	this->cudaStatus = hipGetLastError();
	this->checkCudaError("Launch Kernel", "Device");
	// Unpack yuv key from decklink and store it in yUnpackedCudaKey
	yuyvPackedToyuyvUnpacked <<<grid, block>>>(
				(uint4*)this->pKey,
				this->key,
				srcAlignedWidth,
				dstAlignedWidth,
				this->iHeight
			);
	this->cudaStatus = hipGetLastError();
	this->checkCudaError("Launch Kernel", "Device");
	// Unpack yuv fill from decklink and store it in yUnpackedCudaFill
	yuyvPackedToyuyvUnpacked <<<grid, block>>>(
				(uint4*)this->pFill,
				this->fill,
				srcAlignedWidth,
				dstAlignedWidth,
				this->iHeight
			);
	this->cudaStatus = hipGetLastError();
	this->checkCudaError("Launch Kernel", "Device");

	this->cudaStatus = hipDeviceSynchronize();
	this->checkCudaError("synchronize device", " at unpacking");

	this->cudaStatus = hipMemcpy(this->augVideo, this->video, this->frameSizeUnpacked, hipMemcpyDeviceToDevice);
	this->checkCudaError("copy data", " augmented video buffer");
}



void Preprocessor::create()
{

}

LookupTable::LookupTable(IPipeline *obj): IPipeline(obj)
{
	this->lookupBuffer = nullptr;
	this->loaded = false;
	this->snapShot = nullptr;
}

void LookupTable::create()
{
	this->loaded = false;
}

void LookupTable::update(bool clickEn, MouseData md, std::unordered_map<std::string, int> ws)
{
	if(!clickEn)return;

	if (md.bHandleLDown)
	{
		this->loaded = false;
		int maxRecSize = 200;
		float ScalingValue = maxRecSize*1.0/ws["Outer Diam"]*1.0;

		const dim3 block(16, 16);
		const dim3 grid(
						iDivUp((ws["Outer Diam"]+ws["UV Diam"])*2, block.x),
						iDivUp((ws["Outer Diam"]+ws["UV Diam"])*2, block.y)
						);

		for (int x = (md.iXUpDynamic / 2); x<(md.iXDownDynamic /2); x++)
		{
			for (int y = md.iYUpDynamic; y < md.iYDownDynamic; y=y+2)
			{
				UpdateLookupFrom_XY_Posision_Diffrent_Scaling <<<grid, block>>> (
						this->snapShot,
						this->lookupBuffer,
						x, y,
						(this->iWidth / 2),
						1,10,5,
//						ws["Outer Diam"]*2,
//						ws["UV Diam"]*2,
//						ws["E Lum"],
						ScalingValue,
						maxRecSize
						);
				this->cudaStatus = hipGetLastError();
				this->checkCudaError("Launch kernel", "Device");
			}
		}

		this->cudaStatus = hipDeviceSynchronize();
		this->checkCudaError("synchronize host", " kernel: updateLookupFromMouse");
		assert(this->cudaStatus==hipSuccess);
		this->loaded = true;
	}
}

void LookupTable::clearTable()
{
	this->cudaStatus = hipMemset(this->lookupBuffer, 0, this->iWidth*this->iHeight*sizeof(uchar));
	assert(this->cudaStatus==hipSuccess);
}

void IMask::init()
{

}

void IMask::erode(int size)
{
	cv::cuda::GpuMat chrommaMaskInput;
	cv::cuda::GpuMat chrommaMaskOutput(this->iWidth/2,this->iHeight*2,CV_8UC1, this->maskBuffer,Mat::CONTINUOUS_FLAG);
	chrommaMaskOutput.step=this->iWidth*2;

	// erode output mask
	int an = size;
	cv::Mat element = getStructuringElement(MORPH_ELLIPSE, Size(an*2+1, an*2+1), Point(an, an));
	Ptr<cv::cuda::Filter> erodeFilter = cv::cuda::createMorphologyFilter(MORPH_ERODE, chrommaMaskInput.type(), element);
	erodeFilter->apply(chrommaMaskInput, chrommaMaskOutput);
}

void IMask::dilate(int size)
{
	cv::cuda::GpuMat chrommaMaskInput;
	cv::cuda::GpuMat chrommaMaskOutput(this->iWidth/2,this->iHeight*2,CV_8UC1, this->maskBuffer,Mat::CONTINUOUS_FLAG);
	chrommaMaskOutput.step=this->iWidth*2;

	// Dilate the output mask
	int an = size;
	cv::Mat element = getStructuringElement(MORPH_ELLIPSE, Size(an*2+1, an*2+1), Point(an, an));
	Ptr<cv::cuda::Filter> erodeFilter2 = cv::cuda::createMorphologyFilter(MORPH_DILATE, chrommaMaskInput.type(), element);
	erodeFilter2->apply(chrommaMaskInput, chrommaMaskOutput);
}

ChrommaMask::ChrommaMask(IPipeline* obj, LookupTable* t): IMask(obj)
{
	this->table = t;
}

void ChrommaMask::create()
{
	if(!table->isLoaded())return;
	this->mask = false;
	const int dstAlignedWidth = this->iWidth;
	const int srcAlignedWidth = this->iWidth/2;
	const dim3 block(16, 16);
	const dim3 grid(iDivUp(srcAlignedWidth, block.x), iDivUp(this->iHeight, block.y));

	yuyv_Unpacked_GenerateMask <<<grid, block>>> (
			(uint4*)this->augVideo,
			this->maskBuffer,
			this->table->output(),
			this->iWidth,
			this->iHeight,
			srcAlignedWidth,
			dstAlignedWidth,
			0
			);
	this->cudaStatus = hipGetLastError();
	assert(this->cudaStatus==hipSuccess);

	this->cudaStatus = hipDeviceSynchronize();
	this->checkCudaError("synchronize host", "yuyvGenerateMask");
	assert((this->cudaStatus == hipSuccess));

	this->mask = true;
}

uchar* ChrommaMask::output()
{
	this->create();
	if(!this->mask)return nullptr;
	this->update(); // clean it up and post-process it.
	return this->maskBuffer;
}

void ChrommaMask::toRGB(uchar3* rgb/*Cuda Pointer*/)
{
	const dim3 block(16, 16);
	const dim3 grid(iDivUp(this->iWidth/2, block.x), iDivUp(this->iHeight, block.y));
	const int dstAlignedWidth = this->iWidth;

	Msk2RGB <<<grid, block>>> (
			this->maskBuffer,
			this->maskBuffer,
			this->maskBuffer,
			rgb,
			this->iWidth/2, // source aligned width
			dstAlignedWidth,
			this->iHeight
			);
	this->cudaStatus = hipGetLastError();
	assert(this->cudaStatus==hipSuccess);

	this->cudaStatus = hipDeviceSynchronize();
	assert(this->cudaStatus==hipSuccess);
}

void ChrommaMask::toRGB()
{
	if(this->maskRGB==nullptr) return;
	if(!this->mask) return;
	const dim3 block(16, 16);
	const dim3 grid(iDivUp(this->iWidth/2, block.x), iDivUp(this->iHeight, block.y));
	const int dstAlignedWidth = this->iWidth;

	Msk2RGB <<<grid, block>>> (
			this->maskBuffer,
			this->maskBuffer,
			this->maskBuffer,
			this->maskRGB,
			this->iWidth/2, // source aligned width
			dstAlignedWidth,
			this->iHeight
			);
	this->cudaStatus = hipGetLastError();
	assert(this->cudaStatus==hipSuccess);

	this->cudaStatus = hipDeviceSynchronize();
	assert(this->cudaStatus==hipSuccess);
}

void YoloMask::create()
{

}

uchar* YoloMask::output()
{
	return this->maskBuffer;
}

void MaskOut::create()
{
	// perform some mask creation here ...
}

uchar* MaskOut::output()
{
	return this->chromma->output();
}


Keyer::Keyer(IPipeline* obj, uchar* mask): IPipeline(obj)
{
	this->finalMask = mask;
	this->parabolic = calc_parabola_vertex(0, 0, 512, 1, 1024, 0);
}

void Keyer::create()
{

	const int dstAlignedWidth = (this->iWidth / 2);
	const dim3 block(16, 16);
	const dim3 grid(iDivUp(dstAlignedWidth, block.x), iDivUp(this->iHeight, block.y));
	const int maskWidth = this->iWidth;

	keyAndFill<<<grid, block>>>(
			this->augVideo, // Remember to replace with video after testing...
			this->fill,
			this->key,
			this->iWidth,
			this->iHeight,
			dstAlignedWidth,
			maskWidth,
			this->finalMask,
			480,
			this->parabolic
		);
	this->cudaStatus = hipGetLastError();
	assert(this->cudaStatus==hipSuccess);

	this->cudaStatus = hipDeviceSynchronize();
	assert(this->cudaStatus==hipSuccess);

}



inline void allocateMemory(void** devptr, long int size)
{
	hipError_t cudaStatus = hipMalloc(devptr, size);
	assert(cudaStatus==hipSuccess);
}

void startPipeline()
{
	uchar *chrommaLookupBuffer, *chrommaMask;
	uchar2 *pVideo, *pKey, *pFill;
	uchar3* rgbVideo, *vSnapshot, *maskRGB;
	uint4 *video, *key, *fill, *aVideo, *snapShotV;

	VideoIn decklink;

	Input *in = new Input(&decklink);

	/*************************************************************************************
	 * This is for memory alignment                                                      *
	 * It seems allocating device memory inside an object causes misalignment,           *
	 * Reason:                                                                           *
	 * 	still need to read more and find out why.                                        *
	 * Solution:                                                                         *
	 * 	Declare memory outside and load it inside, but keep the rest of the flow fixed.  *
	 *************************************************************************************/
	allocateMemory((void**)&pVideo,in->getPFrameSize());
	allocateMemory((void**)&pKey, in->getPFrameSize());
	allocateMemory((void**)&pFill, in->getPFrameSize());

	allocateMemory((void**)&video, in->getFrameSize());
	allocateMemory((void**)&key, in->getFrameSize());
	allocateMemory((void**)&fill, in->getFrameSize());
	allocateMemory((void**)&aVideo, in->getFrameSize());
	allocateMemory((void**)&snapShotV, in->getFrameSize());
	allocateMemory((void**)&vSnapshot, in->getWidth()*in->getHeight()*sizeof(uchar3));
	allocateMemory((void**)&rgbVideo, in->getWidth()*in->getHeight()*sizeof(uchar3));
	allocateMemory((void**)&maskRGB, in->getWidth()*in->getHeight()*sizeof(uchar3));

	allocateMemory((void**)&chrommaMask, in->getWidth()*in->getHeight()*sizeof(uchar));
	allocateMemory((void**)&chrommaLookupBuffer, 1024*1024*1024);

	WindowsContainer uiContainer;

	WindowI mainWindow("Main"); // plays the video playback

	KeyingWindow keyingWindow("Keying Window", in->getWidth(), in->getHeight()); // keying window

	SettingsWindow settings("Setting"); // settings

	WindowI maskPreview("Mask Preview");

	keyingWindow.enableMouse();

	uiContainer.addWindow(&mainWindow);
	uiContainer.addWindow(&keyingWindow);
	uiContainer.addWindow(&settings);

	in->load(pVideo, pKey, pFill);

	in->run();

	if(in->isOutput())
	{
		Preprocessor *pp = new Preprocessor(in, in->getPVideo(), in->getPKey(), in->getPFill());
		pp->load(video, key, fill, aVideo, rgbVideo);

		SnapShot *ss = new SnapShot(pp);
		ss->load(vSnapshot, snapShotV);

		Preview *prev = new Preview(ss);
		prev->load(ss->getSnapShot());

		LookupTable *lt = new LookupTable(ss);
		lt->load(chrommaLookupBuffer, snapShotV);

		ChrommaMask *cm = new ChrommaMask(pp, lt);
		cm->load(chrommaMask, maskRGB);

		Keyer *keyer = new Keyer(pp, cm->getMask());

		while(uiContainer.dispatchKey() != 27)
		{
			in->run();
			pp->reload(in->getPVideo(), in->getPKey(), in->getPFill());
			pp->unpack();
			pp->convertToRGB();

			prev->load(pp->getRGB());
			prev->preview(mainWindow.getHandle());

			if(uiContainer.getKey() == 'q')
			{
				ss->takeSnapShot();
				keyingWindow.loadImage(ss->getSnapShot());
				keyingWindow.show();
				cm->output();
				if(cm->isMask())
				{
					cm->toRGB();
					cv::cuda::GpuMat mat;
					mat.create(pp->getHeight(), pp->getWidth(), CV_8UC3);
					mat.step = 5760;
					mat.data = (uchar*)cm->getMaskRGB();

					cv::Mat prev;
					mat.download(prev);
					cv::imshow("Mask Preview", prev);
				}
			}

			if(cm->isMask())
			{
				keyer->create();
				keyer->convertToRGB();
				prev->load(keyer->getRGB());
				prev->preview(mainWindow.getHandle());
			}

			if(keyingWindow.isCaptured())// frame is captured
			{
				lt->update(keyingWindow.isCaptured(), keyingWindow.getMD(), settings.getTrackbarValues());
			}

			keyingWindow.update();
		}
	}

	std::cout<<"Pipeline finished"<<std::endl;
}


