#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : CudaChromaUbuntu.cu
 Author      : Jurie Vosloo
 Version     :
 Copyright   : dont know
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include <npp.h>
#include ""
#include <iostream>
#include <stdio.h>
#include <opencv2/cudacodec.hpp>
#include <iostream>
#include <fstream>
#include <string>
#include <list>
#include "InputLoopThrough.h"
#include "CameraUDP.h"
#include <GL/gl.h>
#include <GL/glu.h>
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <condition_variable>
#include <list>
#include <opencv2/opencv.hpp>
#include <dirent.h> // for linux systems
#include <sys/stat.h> // for linux systems
#include <algorithm>    // std::sort
#include <thread>         // std::this_thread::sleep_for
#include <chrono>         // std::chrono::seconds
#include <iostream>
#include <cstdlib>
#include <pthread.h>
#include <thread>         // std::this_thread::sleep_for
#include <chrono>         // std::chrono::seconds
#include <hip/hip_runtime.h>
#include <math.h>
#include <X11/Xlib.h>
#include "YUVUChroma.cuh"
#include <iostream>       // std::cout
#include "PosisionUpdateUDP.h"

//#include <C:\Users\jurie\Documents\opencv\sources\modules\cudafilters\include\opencv2/cudafilters.hpp>

//#include "DeckLinkAPIVersion.h"



//#define PREVIEW_OUTPUTRENDER
//#define DISPLAY_I_TIMINGS


#define MAX_PATH 260


extern void initPosUDPData();
extern void StartMonitor();
extern void ExitMonitor();
extern bool bGenGenlockStatus();
// Utilities and timing functions
extern bool TestDetectionsPTR(int mousex,int mousey);
extern void initCameraUDPData();
extern void InitResnet18();
extern void DestroyResnet18();
extern int Classify(cv::Mat img_size);
extern int InitYolov5();
extern void CameraZero();
extern float *GetSegmentedMask();
//https://arxiv.org/pdf/1912.05445.pdf
extern int writeframe(cv::Mat frame);


Mutex MouseMutex;
void initOpenCVWindows();
void InitSettingsWindows();

using namespace cv; // all the new API is put into "cv" namespace. Export its content
using namespace std;
using namespace cv::cuda;




bool bExite = false;
bool bClearOutPut =false;
int iExitCount = 0;
bool bBypass = false;
bool bExitWorkerThread = false;
bool bRecordingTrainingData = false;
bool bEnableClick = false;
bool bTrackReset =false;
std::mutex mtxScreenCard;           // mutex for critical section
#define AVG_CALC 25

float m_fNMS=0.45;;
int m_BlendPos = 480;

struct MouseData
{

	int iXDown = 0;
	int iYDown = 0;
	int iXUp = 0;
	int iYUp = 0;
	int iXDownDynamic = 0;
	int iYDownDynamic = 0;
	int iXUpDynamic = 0;
	int iYUpDynamic = 0;
	int x;
	int y;

	bool bHandleLDown = false;
	bool bHandleRDown = false;
	bool bHandleL = false;
	bool bHandleR = false;

};

MouseData MouseData1;
MouseData MouseData2;
MouseData MouseData3;


double4 calc_parabola_vertex(double x1, double y1, double x2, double y2, double x3, double y3)
{
	//http://stackoverflow.com/questions/717762/how-to-calculate-the-vertex-of-a-parabola-given-three-points
	double4 ret;
	double denom = (x1 - x2) * (x1 - x3) * (x2 - x3);
	ret.x = (x3 * (y2 - y1) + x2 * (y1 - y3) + x1 * (y3 - y2)) / denom;
	ret.y = (x3*x3 * (y1 - y2) + x2 * x2 * (y3 - y1) + x1 * x1 * (y2 - y3)) / denom;
	ret.z = (x2 * x3 * (x2 - x3) * y1 + x3 * x1 * (x3 - x1) * y2 + x1 * x2 * (x1 - x2) * y3) / denom;
	if (x1 == 0 && x3 == 1024)
		ret.w = 0;
	else
		ret.w = 1;

	return ret;

}

struct WindowSettings
{
	int m_iUV_Diam;
	int m_iLum_Diam;
	int m_iOuter_Diam ;
	int m_iErase_Diam ;
	int m_iErase_Lum_Diam ;

	int m_iErode;
	int m_iDilate ;
	int m_iLowerlimit ;
	int m_iUpperlimit ;

	int m_cunnyb;

	int m_cunnyt;
	double4 m_ParabolicFunc;

	public:
		WindowSettings()
		{
			m_iUV_Diam=4;
			m_iLum_Diam=2;
			m_iOuter_Diam=14;
			m_iErase_Diam=15;
			m_iErase_Lum_Diam=15;
			 m_cunnyb=125;
			 m_cunnyt=274;
			m_iErode=2;
			m_iDilate=1;


			//m_BlendPos = 0;
			m_iLowerlimit = 80;
			m_iUpperlimit=80;
		}


};
int iUpdateIndex = 0;
WindowSettings FourSettings[3];
//FourSettings[0].m_BlendPos=877;

void  CallThisMouse_Masks(int event, int x, int y, int flags, void* userdata)
{
	switch (event)
	{
	case EVENT_LBUTTONDOWN:
	case EVENT_RBUTTONDOWN:
		bool *ptr = (bool *)userdata;
		*ptr = false;

		break;
	}


}

void  CallThisMouseUpDown(int event, int x, int y, int flags, void* userdata)
{
	switch (event)
	{
	case EVENT_LBUTTONDOWN:
		MouseData2.iXDown = x;
		MouseData2.iYDown = y;

		MouseData2.bHandleLDown = true;
		break;

	case EVENT_RBUTTONDOWN:

		MouseData2.iXDown = x;
		MouseData2.iYDown = y;
		MouseData2.bHandleRDown = true;
		break;

	case EVENT_LBUTTONUP:
		MouseData2.bHandleLDown = false;
		//	bHandleL = true;
		MouseData2.iXUp = x;
		MouseData2.iYUp = y;

		break;
	case EVENT_RBUTTONUP:
		MouseData2.bHandleRDown = false;
		//	bHandleR = true;
		MouseData2.iXUp = x;
		MouseData2.iYUp = y;

		break;


	case EVENT_MOUSEMOVE:
	{

		MouseData2.iXUpDynamic = x - 10;
		MouseData2.iYUpDynamic = y - 10;
		MouseData2.iXDownDynamic = x + 10;
		MouseData2.iYDownDynamic = y + 10;

	}

	break;
	}
}

void  MouseUV_FRAME_INFO(int event, int x, int y, int flags, void* userdata)
{
	switch (event)
	{
	case EVENT_LBUTTONDOWN:
		MouseData3.iXDown = x;
		MouseData3.iYDown = y;

		break;

	case EVENT_RBUTTONDOWN:

		MouseData3.iXDown = x;
		MouseData3.iYDown = y;

		break;

	case EVENT_LBUTTONUP:
		MouseData3.bHandleLDown = true;
		//	bHandleL = true;
		MouseData3.iXUp = x;
		MouseData3.iYUp = y;

		break;
	case EVENT_RBUTTONUP:
		MouseData3.bHandleRDown = true;
		//	bHandleR = true;
		MouseData3.iXUp = x;
		MouseData3.iYUp = y;

		break;


	case EVENT_MOUSEMOVE:
		//	if (bHandleRDown || bHandleLDown)
	{

		MouseData3.iXUpDynamic = x - 10;
		MouseData3.iYUpDynamic = y - 10;
		MouseData3.iXDownDynamic = x + 10;
		MouseData3.iYDownDynamic = y + 10;

	}

	break;
	}



}
 int iRecsize = 4;
void  CallThisMouse(int event, int x, int y, int flags, void* userdata)
{
	//Rect t= getWindowImageRect("RGB Output");


	MouseMutex.lock();


	Rect tt = getWindowImageRect("RGB Output");
//	std::cout<<"I execute"<<std::endl;
	//printf("\n\r%d %d %d %d %d %d\n", tt.x, tt.y, tt.width, tt.height,x,y);
	double x1 = double(x)/(double)(tt.width)  * 1920.0;//window correction
	double y1 = double(y)/(double)(tt.height) * 1080.0;//window correction
//	printf("%d %d %d %d %f %f\n\r", tt.x, tt.y, tt.width, tt.height, x1, y1);

	switch (event)
	{


	case EVENT_MOUSEWHEEL ://!< positive and negative values mean forward and backward scrolling, respectively.
		if (flags > 0)
		{
			if(iRecsize>1)
				iRecsize--;
		}
		else
		{
			iRecsize++;

			if (iRecsize > 20)
				iRecsize = 20;
		}

		MouseData1.iXUpDynamic = x1 - iRecsize;
		MouseData1.iYUpDynamic = y1 - iRecsize + 4;
		MouseData1.iXDownDynamic = x1 + iRecsize;
		MouseData1.iYDownDynamic = y1 + iRecsize + 4;

		break;


	case EVENT_LBUTTONDOWN:

		TestDetectionsPTR(x1,y1);
		MouseData1.iXDown = x1;
		MouseData1.iYDown = y1;
		MouseData1.iXUpDynamic = x1 - iRecsize;
		MouseData1.iYUpDynamic = y1 - iRecsize + 4;
		MouseData1.iXDownDynamic = x1 + iRecsize;
		MouseData1.iYDownDynamic = y1 + iRecsize + 4;
		MouseData1.bHandleLDown = true;
		//std::cout << MouseData1.iXUpDynamic;
		 break;

	case EVENT_RBUTTONDOWN:

		MouseData1.iXDown = x1;
		MouseData1.iYDown = y1;
		MouseData1.bHandleRDown = true;
		break;

	case EVENT_LBUTTONUP:
		MouseData1.bHandleLDown = false;
	//	bHandleL = true;
		MouseData1.iXUp = x1;
		MouseData1.iYUp = y1;

		break;
	case EVENT_RBUTTONUP:
		MouseData1.bHandleRDown = false;
	//	bHandleR = true;
		MouseData1.iXUp = x1;
		MouseData1.iYUp = y1;

		break;


	case EVENT_MOUSEMOVE:

	//	if (bHandleRDown || bHandleLDown)
		{
			MouseData1.iXUpDynamic = x1-iRecsize;
			MouseData1.iYUpDynamic = y1- iRecsize+4;
			MouseData1.iXDownDynamic = x1+ iRecsize;
			MouseData1.iYDownDynamic = y1+ iRecsize+4;

		}

		MouseData1.x=x1;
		MouseData1.y=y1;

		break;
	}

	MouseMutex.unlock();
}



bool EveryFrame_L = false;
unsigned int iDelayFrames =1;
int iVrArCut = 795;// - 64
int iVrArCut0 = 300;
int iVrArCut1 = 200;
int iVrArCut2 = 100;
bool DisableParabolicKeying = false;
int iAVGCutOff;

void Blending(int pos, void* userdata)
{
	m_BlendPos = pos;
}

void FrameDelay(int pos, void* userdata)
{
	iDelayFrames = pos;
}

void Erode(int pos, void* userdata)
{
	FourSettings[iUpdateIndex].m_iErode = pos;
}

void Dilate(int pos, void* userdata)
{
	FourSettings[iUpdateIndex].m_iDilate = pos;
}




void UV_DIAMETER(int pos, void* userdata)
{
	FourSettings[iUpdateIndex].m_iUV_Diam = pos;
}

void LUM_DIAM(int pos, void* userdata)
{
	FourSettings[iUpdateIndex].m_iLum_Diam = pos;
}

void OUTER_DIAM(int pos, void* userdata)
{
	FourSettings[iUpdateIndex].m_iOuter_Diam = pos;
}

void ERASE_UV_DIAMETER(int pos, void* userdata)
{
	FourSettings[iUpdateIndex].m_iErase_Diam = pos;
}

void ERASE_LUM_DIAM(int pos, void* userdata)
{
	FourSettings[iUpdateIndex].m_iErase_Lum_Diam = pos;
}

void CUNNY_TOP(int pos, void* userdata)
{
	FourSettings[iUpdateIndex].m_cunnyt=pos;
}
void NMS(int pos, void* userdata)
{
	m_fNMS=float(pos/100.0);
}

void CUNNY_BOT(int pos, void* userdata)
{
	FourSettings[iUpdateIndex].m_cunnyb=pos;

}
void LUM_CUT_BOT(int pos, void* userdata)
{
	FourSettings[iUpdateIndex].m_iLowerlimit = pos;
	FourSettings[iUpdateIndex].m_ParabolicFunc = calc_parabola_vertex(FourSettings[iUpdateIndex].m_iLowerlimit, 0, 512, 1, 1024- FourSettings[iUpdateIndex].m_iUpperlimit, 0);
}

void LUM_CUT_TOP(int pos, void* userdata)
{
	FourSettings[iUpdateIndex].m_iUpperlimit = pos;
	FourSettings[iUpdateIndex].m_ParabolicFunc = calc_parabola_vertex(FourSettings[iUpdateIndex].m_iLowerlimit, 0, 512, 1, 1024 - FourSettings[iUpdateIndex].m_iUpperlimit, 0);
}

struct Traininginfo
{
	Traininginfo()
	{
		DataL.w = -1;
		DataL.x = -1;
		DataL.y = -1;
		DataL.z = -1;
		DataR.w = -1;
		DataR.x = -1;
		DataR.y = -1;
		DataR.z = -1;
	}
	float4 DataL;
	float4 DataR;
};


std::list<Traininginfo*> TrainingDataList;


void DumpTraingData()
{

	char filename[200];
	sprintf(filename, "d:\\TrainingData%d.try",(int) TrainingDataList.size());
	ofstream myfile(filename, ios::binary);

	while (TrainingDataList.size())
	{
		Traininginfo* ptr = TrainingDataList.back();
		TrainingDataList.pop_back();
		myfile.write((char*)ptr, sizeof(Traininginfo));
	}
}


bool bSmall = false;
bool bTakeMask = true;
int bTakeOutput = -1;
bool bDoPaintBack=false;

int iLastCheck = 0;
bool bSafeSnapshot = false;

bool bTaining = false;
bool bErase = false;

int fpsCount = 0;  // FPS count for averaging
int fpsLimit = 1;  // FPS limit for sampling
unsigned int frameCount = 0;
const char *sSDKsample = "RGB Output";

struct ThreadData
{
	ThreadData()
	{
		bUpdateRGB_Preview = false;
		RGB_Output_Cuda=0;
		MouseData1=0;
	}
	cuda::GpuMat *RGB_Output_Cuda;
	bool bUpdateRGB_Preview;
	MouseData *MouseData1;
};

template<typename T>
std::string toString(const T &t) {
    std::ostringstream oss;
    oss << t;
    return oss.str();
}

void SaveImageThread(cv::Mat RGBImageFull,int iIndex,std::string FileName)
{

	int iOffset=0;
	switch(iIndex)
	{

	case 0:
		iOffset=0;
		FileName=FileName+"_0.bmp";
			break;
	case 1:
		iOffset=427;
		FileName=FileName+"_1.bmp";
				break;

	case 2:
		iOffset=853;
		FileName=FileName+"_2.bmp";
			break;


	case 3:
		iOffset=1280;
		FileName=FileName+"_3.bmp";
			break;
	}
	cv::Rect roi(iOffset,0,640,1080/2);
	RGBImageFull.cols=1920*2;
	RGBImageFull.rows=1080/2;
	RGBImageFull.step=1920*2*3;

	cv::Mat croppedImage = RGBImageFull(roi);
	cv::imwrite(FileName,croppedImage);
}

ThreadData myThreadData;
char OutputRenderthreadStatus[MAX_PATH];


void DrawOutputThreadData(Mat *DrawingMat)
{
	cv::Rect r(50,1080-100,10,10);
	cv::rectangle(*DrawingMat,r,Scalar(255,255,255),2);
	std::string text = OutputRenderthreadStatus;
	Mat img=*DrawingMat;

	int fontFace = FONT_HERSHEY_PLAIN;
	double fontScale =2.0;
	int thickness = 2;

	int baseline=0;
	Size textSize = getTextSize(text, fontFace,
					fontScale, thickness, &baseline);
	baseline += thickness;
	// center the text
	Point textOrg(r.x+r.width,
	  r.y/*+r.height/2*/);


	putText(img, text, textOrg, fontFace, fontScale,
		Scalar::all(255), thickness, 8);
}



bool bAutoTrain=false;
//
void *OutputRenderthread(void *lpParam)//https://developer.nvidia.com/blog/this-ai-can-automatically-remove-the-background-from-a-photo/
{

#ifdef PREVIEW_OUTPUTRENDER
	initOpenCVWindows();
	namedWindow("y_only",WINDOW_OPENGL);
	namedWindow("u_only",WINDOW_OPENGL);
	namedWindow("v_only",WINDOW_OPENGL);
	namedWindow("mask Erode dilate GaussianFilter",WINDOW_OPENGL);
	namedWindow("UI Mouse training Window",WINDOW_OPENGL);
	namedWindow("Yolo generated mask",WINDOW_OPENGL);
	namedWindow("Yolo generated mask HistoGram",WINDOW_OPENGL);
	namedWindow("Yolomask",WINDOW_OPENGL);



	int yspace = 1;
	int xspace = 380;
	int index = 0 ;
	moveWindow("y_only",0,0);
	moveWindow("mask Erode dilate GaussianFilter",0*xspace,0*yspace);
	moveWindow("UI Mouse training Window",1*xspace,1*yspace);
	moveWindow("Yolo generated mask",2*xspace,2*yspace);
	moveWindow("u_only",3*xspace,3*yspace);
	moveWindow("v_only",4*xspace,4*yspace);

#endif
//
	bool bPopFront=false;
	unsigned int iDelayFramesStore=1;
	long framecounter=0;

	double avg_duration[AVG_CALC];
	double avg=0.0;
	int iAvgIndex=0;
	ThreadData *ptrThreadData=(ThreadData *)lpParam;
	InitYolov5();

	unsigned int Max_duration=0;
	VideoIn decklink_video_in;
	while (decklink_video_in.m_sizeOfFrame== -1)
	{
		std::this_thread::sleep_for(std::chrono::milliseconds(40));
	}

	CudaChromaInit(decklink_video_in.m_iWidth, decklink_video_in.m_iHeight,decklink_video_in.m_sizeOfFrame, decklink_video_in.m_iFrameSizeUnpacked);
	cudaLookUpInit();
	PrepareYoloData(false,0.9);
	FourSettings[0].m_ParabolicFunc = calc_parabola_vertex(0, 0, 512, 1, 1024, 0);
	FourSettings[1].m_ParabolicFunc = calc_parabola_vertex(0, 0, 512, 1, 1024, 0);
	FourSettings[2].m_ParabolicFunc = calc_parabola_vertex(0, 0, 512, 1, 1024, 0);


//	while (!bExitWorkerThread)
//			{
//
//					//std::this_thread::sleep_for(std::chrono::milliseconds(1));
//					decklink_video_in.WaitForFrames();
//					void * ptr_BG_Video = decklink_video_in.imagelistVideo.GetFrame(true);
//					void * ptr__FILL_Video = decklink_video_in.imagelistFill.GetFrame(true);
//					void * ptr__KEY_Video = decklink_video_in.imagelistKey.GetFrame(true);
//					free(ptr_BG_Video);
//					free(ptr__FILL_Video);
//					free(ptr__KEY_Video);
//			}

	decklink_video_in.WaitForFrames(1);
	decklink_video_in.imagelistVideo.ClearAll(0);
	decklink_video_in.imagelistFill.ClearAll(0);
	decklink_video_in.imagelistKey.ClearAll(0);
	decklink_video_in.ImagelistOutput.ClearAll(1);


		while (!bExitWorkerThread)
		{




		//	std::this_thread::sleep_for(std::chrono::milliseconds(10));
			auto timer_wait_start = std::chrono::system_clock::now();
		if((iDelayFramesStore!=iDelayFrames)||bClearOutPut)
		{
			bClearOutPut=false;
			iDelayFramesStore=iDelayFrames;
			decklink_video_in.WaitForFrames(-1);
			decklink_video_in.ImagelistOutput.ClearAll(1);
			decklink_video_in.imagelistVideo.ClearAll(0);
			decklink_video_in.imagelistFill.ClearAll(0);
			decklink_video_in.imagelistKey.ClearAll(0);
		}else
			decklink_video_in.WaitForFrames(iDelayFrames);
			auto timer_start = std::chrono::system_clock::now();
			avg=0.0;
			for(int x=0;x<AVG_CALC;x++)
				avg=avg+avg_duration[x];
			avg=avg/AVG_CALC;
			snprintf(OutputRenderthreadStatus,sizeof(OutputRenderthreadStatus),"avg:%f,Genlocked:%s Video:%d Key:%d Fill:%d Delay:%d Output:%d \r",avg, bGenGenlockStatus() ? "Yes" : "No",(int) decklink_video_in.imagelistVideo.GetFrameCount(),  (int)(int) decklink_video_in.imagelistKey.GetFrameCount(), (int)(int) decklink_video_in.imagelistFill.GetFrameCount(), iDelayFrames, (int)(int) decklink_video_in.ImagelistOutput.GetFrameCount());
			framecounter++;
			unsigned int iBufferCount=decklink_video_in.imagelistVideo.GetFrameCount() ;
			if(iDelayFrames<iBufferCount)
			{
				bPopFront=true;
			}
			else
			{
				bPopFront=false;
			}
			void * ptr_BG_Video = decklink_video_in.imagelistVideo.GetFrame(bPopFront);
			void * ptr__FILL_Video = decklink_video_in.imagelistFill.GetFrame(true);
			void * ptr__KEY_Video = decklink_video_in.imagelistKey.GetFrame(true);
			mtxScreenCard.lock();

			CudaSetInputData(ptr_BG_Video,ptr__FILL_Video,ptr__KEY_Video,false);
			auto  timer_start_CudaSetInputData = std::chrono::system_clock::now();
			if(bPopFront)
				free(ptr_BG_Video);
			free(ptr__FILL_Video);
			free(ptr__KEY_Video);

#ifdef PREVIEW_OUTPUTRENDER
			Launch_yuyv10PackedToyuyvUnpacked(decklink_video_in.m_RowLength,bTakeMask,decklink_video_in.m_iFrameSizeUnpacked,ptrThreadData->RGB_Output_Cuda,FourSettings[0].m_cunnyb,FourSettings[0].m_cunnyt,true);
#else
			Launch_yuyv10PackedToyuyvUnpacked(decklink_video_in.m_RowLength,bTakeMask,decklink_video_in.m_iFrameSizeUnpacked,ptrThreadData->RGB_Output_Cuda,FourSettings[0].m_cunnyb,FourSettings[0].m_cunnyt,bAutoTrain);
			#endif
			auto timer_Launch_yuyv10PackedToyuyvUnpacked = std::chrono::system_clock::now();



		//auto startyolo = std::chrono::system_clock::now();
		if(bTakeOutput==-1)
			PrepareYoloData(bTakeMask,m_fNMS);
		else
			PrepareYoloData(true,m_fNMS);
		auto timer_endyolo = std::chrono::system_clock::now();

		if (bTakeMask)
		{
			bTakeMask = false;
			ptrThreadData->bUpdateRGB_Preview = true;
		}

		#ifdef PREVIEW_OUTPUTRENDER
			Launch_yuyv_Unpacked_GenerateMask(0, x,true);//
		`	Launch_yuyvDilateAndErode(FourSettings[x].m_iErode, FourSettings[x].m_iDilate, x);
		#else
			Launch_yuyv_Unpacked_GenerateMask(0, 0,bAutoTrain);//
			Launch_yuyvDilateAndErode(FourSettings[0].m_iErode, FourSettings[0].m_iDilate, 0);
			Launch_yuyv_Unpacked_GenerateMask_yolo_seg(0, 1,bAutoTrain,GetSegmentedMask());//
			//Launch_yuyvDilateAndErode(FourSettings[1].m_iErode, FourSettings[1].m_iDilate, 1);

		#endif

		auto timer_Launch_yuyvDilateAndErode_Launch_yuyv_Unpacked_GenerateMask_yolo = std::chrono::system_clock::now();
		Launch_yuyv_Unpacked_UnpackedComBineData(&m_BlendPos, &m_BlendPos, &m_BlendPos, decklink_video_in.m_RowLength, &FourSettings[0].m_ParabolicFunc, &FourSettings[1].m_ParabolicFunc, &FourSettings[2].m_ParabolicFunc,bBypass, iVrArCut+ 64, iVrArCut0+64, iVrArCut1 + 64, iVrArCut2 + 64, bTakeOutput);
		auto timer_Launch_yuyv_Unpacked_UnpackedComBineData = std::chrono::system_clock::now();
		if (bTakeOutput!=-1)
		{
			ptrThreadData->bUpdateRGB_Preview = true;
			//bTakeOutput = -1;
		}


		void *yuvdata = malloc(decklink_video_in.m_sizeOfFrame);
		CudaGetOutputData(yuvdata);
		decklink_video_in.ImagelistOutput.AddFrame(yuvdata);
		auto timer_end = std::chrono::system_clock::now();
		mtxScreenCard.unlock();
		auto duration_now=std::chrono::duration_cast<std::chrono::milliseconds>(timer_end - timer_start).count();
	//	auto duration_now=std::chrono::duration_cast<std::chrono::milliseconds>(timer_end - timer_start).count();
		auto duration_yolo=std::chrono::duration_cast<std::chrono::milliseconds>(timer_endyolo - timer_Launch_yuyv10PackedToyuyvUnpacked).count();

		auto timer_start_wait_duration=std::chrono::duration_cast<std::chrono::milliseconds>(timer_start-timer_wait_start ).count();//
		avg_duration[iAvgIndex++]=duration_now;
		if(iAvgIndex==AVG_CALC)
			iAvgIndex=0;
		if(Max_duration<duration_now)
			Max_duration=duration_now;

#ifdef DISPLAY_I_TIMINGS
		if((framecounter==10))
		{
			auto duration_CudaSetInputData				=std::chrono::duration_cast<std::chrono::milliseconds>(timer_start-timer_start_CudaSetInputData).count();
			auto duration_yuyv10PackedToyuyvUnpacked	=std::chrono::duration_cast<std::chrono::microseconds>(timer_start_CudaSetInputData-timer_Launch_yuyv10PackedToyuyvUnpacked).count();
			auto duration_PrepareYoloData				=std::chrono::duration_cast<std::chrono::milliseconds>(timer_Launch_yuyv10PackedToyuyvUnpacked-timer_endyolo).count();
			auto duration_generatemask_erode_dilate		=std::chrono::duration_cast<std::chrono::milliseconds>(timer_endyolo-timer_Launch_yuyvDilateAndErode_Launch_yuyv_Unpacked_GenerateMask_yolo).count();
			auto duration_add_to_decklink				=std::chrono::duration_cast<std::chrono::milliseconds>(timer_Launch_yuyv_Unpacked_UnpackedComBineData-timer_end).count();
			std::cout <<"duration_CudaSetInputData"<<duration_CudaSetInputData<< "ms  "<<std::endl;
			std::cout <<"duration_yuyv10PackedToyuyvUnpacked"<<duration_yuyv10PackedToyuyvUnpacked<< "us  "<<std::endl;
			std::cout <<"duration_PrepareYoloData"<<duration_PrepareYoloData<< "ms  "<<std::endl;
			std::cout <<"duration_generatemask_erode_dilate"<<duration_generatemask_erode_dilate<< "ms  "<<std::endl;
			std::cout <<"duration_add_to_decklink"<<duration_add_to_decklink<< "ms  "<<std::endl;


			framecounter=0;
			std::cout <<"Max:"<<Max_duration<< "ms Now:"<< duration_now << "ms  " <<std::endl;
			Max_duration=0;

		}
#endif


		}
		std::cout <<"Launch_yuyv10PackedToyuyvUnpacked"<<std::endl;
		CudaChromaFree();
		cudaLookUpFree();
		return 0;


}


void ButtonCallback_KeyTopAndBottom(int state, void* userdata)
{





}




void InitSettingsWindows()
{
	namedWindow("Settings", WINDOW_NORMAL);
	//createTrackbar("Blending", "Settings", 0, 877, Blending, 0);
	createTrackbar("Blending", "Settings", 0, 2000, Blending, 0);
	createTrackbar("Delay", "Settings", 0, 30, FrameDelay, 0);
	createTrackbar("Erode", "Settings", 0, 20, Erode, 0);
	createTrackbar("Dialate", "Settings", 0, 20, Dilate, 0);
	createTrackbar("Outer Diam", "Settings", 0, 200, OUTER_DIAM, 0);
	createTrackbar("UV Diam", "Settings", 0, 50, UV_DIAMETER, 0);
	createTrackbar("Lum Depth", "Settings", 0, 50, LUM_DIAM, 0);
	createTrackbar("E UV", "Settings", 0, 50, ERASE_UV_DIAMETER, 0);
	createTrackbar("E Lum", "Settings", 0, 50, ERASE_LUM_DIAM, 0);
	createTrackbar("Key Bot", "Settings", 0, 300, LUM_CUT_BOT, 0);
	createTrackbar("Key Top", "Settings", 0, 300, LUM_CUT_TOP, 0);

//	createTrackbar("AR/VR Cut", "Settings", &iVrArCut, (877),0, 0);
//	createTrackbar("VR Cut 1", "Settings", &iVrArCut0, (877), 0, 0);
//	createTrackbar("VR Cut 2", "Settings", &iVrArCut1, (877), 0, 0);
//	createTrackbar("VR Cut 3", "Settings", &iVrArCut2, (877), 0, 0);
//	setTrackbarPos("AR/VR Cut", "Settings", 877);

//	createTrackbar("canny L", "Settings", 0, 900, CUNNY_BOT, 0);
//	createTrackbar("canny T", "Settings", 0, 900, CUNNY_TOP, 0);
	createTrackbar("NMS", "Settings", 0, 100, NMS, 0);
	//createButton("Enable Key Top And Bottom,", ButtonCallback_KeyTopAndBottom, 0, QT_CHECKBOX, true);
}


void UpdateSettingsWindow()
{
	setTrackbarPos("Blending", "Settings", m_BlendPos);
	setTrackbarPos("Delay", "Settings", 3);

	setTrackbarPos("UV Diam", "Settings", FourSettings[iUpdateIndex].m_iUV_Diam);
	setTrackbarPos("Outer Diam", "Settings", FourSettings[iUpdateIndex].m_iOuter_Diam);
	setTrackbarPos("Lum Depth", "Settings", FourSettings[iUpdateIndex].m_iLum_Diam);
	setTrackbarPos("E UV", "Settings", FourSettings[iUpdateIndex].m_iErase_Diam);
	setTrackbarPos("E Lum", "Settings", FourSettings[iUpdateIndex].m_iErase_Lum_Diam);
	setTrackbarPos("Erode", "Settings", FourSettings[iUpdateIndex].m_iErode);
	setTrackbarPos("Dialate", "Settings", FourSettings[iUpdateIndex].m_iDilate);
	setTrackbarPos("Key Bot", "Settings", FourSettings[iUpdateIndex].m_iLowerlimit);
	setTrackbarPos("Key Top", "Settings", FourSettings[iUpdateIndex].m_iUpperlimit);
	//setTrackbarPos("canny L", "Settings", FourSettings[iUpdateIndex].m_cunnyb);
	//setTrackbarPos("canny T", "Settings", FourSettings[iUpdateIndex].m_cunnyt);
	setTrackbarPos("NMS", "Settings", m_fNMS*100);

}

void on_opengl(void* param)
{

	glViewport(0, 0,  1366.0, 768.0);
	glMatrixMode(GL_PROJECTION);                        // Select The Projection Matrix
	glLoadIdentity();                           // Reset The Projection Matrix

	// Calculate The Aspect Ratio Of The Window
	//	if(m_iWindowX>m_iWindowY)
	gluOrtho2D(0, 1366.0, 768.0, 0);

	//	gluPerspective(45.0f,(GLfloat)m_iWindowX/(GLfloat)m_iWindowY,0.1f,100.0f);

	glMatrixMode(GL_MODELVIEW);                     // Select The Modelview Matrix
	glLoadIdentity();                           // Reset The Modelview Matrix
	/*glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho (0,1920,0,1080,-1, 1);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();*/
	//	glEnable(GL_TEXTURE_2D);                        // Enable Texture Mapping ( NEW )
	glEnable(GL_BLEND);
	glShadeModel(GL_SMOOTH);                        // Enable Smooth Shading
	glClearDepth(1.0f);                         // Depth Buffer Setup
	glEnable(GL_DEPTH_TEST);                        // Enables Depth Testing
	glDepthFunc(GL_ALWAYS);                         // The Type Of Depth Testing To Do
	glHint(GL_PERSPECTIVE_CORRECTION_HINT, GL_NICEST);          // Really Nice Perspective
	glDisable(GL_LIGHTING);
	glEnable(GL_MULTISAMPLE_ARB);
	//	glClear(
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glColor4f(GLfloat(40.0 / 255.0), GLfloat(40.0 / 255.0), GLfloat(40.0 / 255.0), GLfloat(1.0));
	glBegin(GL_QUADS);
	glVertex2f(10, 10);
	glVertex2f(1910, 10);
	glVertex2f(1910, 1070);
	glVertex2f(10, 1070);
	glVertex2f(10, 10);
	glEnd();
	glColor4f(GLfloat(1), GLfloat(1), GLfloat(1), GLfloat(1.0));
	glBegin(GL_LINE_LOOP);
	glVertex2f(10, 10);
	glVertex2f(1910, 10);
	glVertex2f(1910, 1070);
	glVertex2f(10, 1070);
	glVertex2f(10, 10);
	glEnd();
	glFlush();

//	std::cout<<"Finished Setting the window"<<std::endl;


//	 glClear(GL_COLOR_BUFFER_BIT);
//
//	  // Draw a white grid "floor" for the tetrahedron to sit on.
//	  glColor3f(1.0, 1.0, 1.0);
//	  glBegin(GL_LINES);
//	  for (GLfloat i = -2.5; i <= 2.5; i += 0.25) {
//	    glVertex3f(i, 0, 2.5); glVertex3f(i, 0, -2.5);
//	    glVertex3f(2.5, 0, i); glVertex3f(-2.5, 0, i);
//	  }
//	  glEnd();
//
//	  // Draw the tetrahedron.  It is a four sided figure, so when defining it
//	  // with a triangle strip we have to repeat the last two vertices.
//	  glBegin(GL_TRIANGLE_STRIP);
//	    glColor3f(1, 1, 1); glVertex3f(0, 2, 0);
//	    glColor3f(1, 0, 0); glVertex3f(-1, 0, 1);
//	    glColor3f(0, 1, 0); glVertex3f(1, 0, 1);
//	    glColor3f(0, 0, 1); glVertex3f(0, 0, -1.4);
//	    glColor3f(1, 1, 1); glVertex3f(0, 2, 0);
//	    glColor3f(1, 0, 0); glVertex3f(-1, 0, 1);
//	  glEnd();
//
//	  glFlush();

}

void initOpenCVWindows()
{

	InitSettingsWindows();
	UpdateSettingsWindow();
	std::string rgbOutputWin = "RGB Output";
	std::string frameInfoWin = "Frame Info";

	namedWindow(rgbOutputWin,  WINDOW_NORMAL);
	//namedWindow("RGB Output1",  WINDOW_OPENGL);
	namedWindow(frameInfoWin, WINDOW_NORMAL);
//	namedWindow("Soccer Frame Info", WINDOW_OPENGL);
	setMouseCallback(rgbOutputWin, CallThisMouse, 0);
//	setOpenGlDrawCallback(rgbOutputWin, on_opengl, 0);
	updateWindow(rgbOutputWin);

//	std::cout<<"I run"<<std::endl;

}

void UpdateTXTFile(char *Buffer)
{
	std::ofstream outfile(GetTXTFileName(), std::ofstream::binary| std::ofstream::app);
	outfile.write(Buffer, strlen(Buffer));
}

void UpdateLookupFromMouse()
{

	if (bTaining || bErase)
	{
	/*	MASK->setTo(0);

		if (bTaining)
			rectangle(*MASK, Rect(Point(MouseData1.iXDown, MouseData1.iYDown), Point(MouseData1.iXUp, MouseData1.iYUp)), Scalar(255), CV_FILLED);
		if (bErase)
			rectangle(*MASK, Rect(Point(MouseData1.iXDown, MouseData1.iYDown), Point(MouseData1.iXUp, MouseData1.iYUp)), Scalar(128), CV_FILLED);

		return true;*/
	}
	else
	{

		if (bEnableClick)
		if (MouseData1.bHandleLDown)
		{

			//MASK->setTo(0);
			//rectangle(*MASK, Rect(Point(MouseData1.iXDownDynamic, MouseData1.iYDownDynamic), Point(MouseData1.iXUpDynamic, MouseData1.iYUpDynamic)), Scalar(255), CV_FILLED);

			if (!bSmall)
			{
				char buffer[MAX_PATH];
				sprintf(buffer, "1 %d %d %d %d %d\n", MouseData1.iXUpDynamic, MouseData1.iYUpDynamic, MouseData1.iXDownDynamic, MouseData1.iYDownDynamic,FourSettings[iUpdateIndex].m_iUV_Diam);
				UpdateTXTFile(buffer);
				//std::cout << buffer << std::endl;
				mtxScreenCard.lock();
				Launch_UpdateLookupFrom_XY_Posision(MouseData1.iXUpDynamic, MouseData1.iYUpDynamic, MouseData1.iXDownDynamic, MouseData1.iYDownDynamic, FourSettings[iUpdateIndex].m_iUV_Diam, FourSettings[iUpdateIndex].m_iLum_Diam, FourSettings[iUpdateIndex].m_iOuter_Diam, 200,bDoPaintBack);
				mtxScreenCard.unlock();

			}
			else
			{
				char buffer[MAX_PATH];
				sprintf(buffer, "1 %d %d %d %d\n", MouseData1.iXUpDynamic, MouseData1.iYUpDynamic, MouseData1.iXDownDynamic, MouseData1.iYDownDynamic);
				UpdateTXTFile(buffer);
				mtxScreenCard.lock();
				Launch_UpdateLookupFrom_XY_Posision(MouseData1.iXUpDynamic, MouseData1.iYUpDynamic, MouseData1.iXDownDynamic, MouseData1.iYDownDynamic, 1, 10, 5, 200,bDoPaintBack);
				mtxScreenCard.unlock();
			}
			return;
		}
		if (bEnableClick)
		if (MouseData1.bHandleRDown)
		{
			char buffer[MAX_PATH];
			sprintf(buffer, "0 %d %d %d %d\n", MouseData1.iXUpDynamic, MouseData1.iYUpDynamic, MouseData1.iXDownDynamic, MouseData1.iYDownDynamic);
			UpdateTXTFile(buffer);
			mtxScreenCard.lock();
			Launch_UpdateLookupFrom_XY_Posision_Erase(MouseData1.iXUpDynamic, MouseData1.iYUpDynamic, MouseData1.iXDownDynamic, MouseData1.iYDownDynamic, FourSettings[iUpdateIndex].m_iErase_Diam, FourSettings[iUpdateIndex].m_iErase_Lum_Diam,bDoPaintBack);
			mtxScreenCard.unlock();
		//	return;
		}
	}




}

#define VK_LCONTROL 0
#define VK_F1 190
#define VK_F2 191
#define VK_F3 192
#define VK_F4 193
#define VK_F5 194

#define VK_F10 199
int iKey;
void UpdateKeyState()
{
	iKey=waitKey(10);
//	while(waitKey(1)!=-1);
//	if(iKey!=-1)
	//	std::cout << iKey << std ::  endl;
//	std::this_thread::sleep_for(std::chrono::milliseconds(1));
}


bool GetAsyncKeyState(int checkpressed)
{
	if(checkpressed==iKey)
	{

		iKey=-1;
		return true;
	}
	else
		return false;

	return false;
}


void DrawMouseText(Mat *DrawingMat,string text,cv::Point r)
{


	char buffer[MAX_PATH];
	sprintf(buffer, "%d %d\n", MouseData1.x,MouseData1.y);


	Mat img=*DrawingMat;

	int fontFace = FONT_HERSHEY_PLAIN;
	double fontScale =1.0;
	int thickness = 1;

	int baseline=0;
//	Size textSize = getTextSize(text, fontFace,
//								fontScale, thickness, &baseline);
//	baseline += thickness;
	// center the text
	Point textOrg(r.x,r.y);


	putText(img, buffer, textOrg, fontFace, fontScale,
			Scalar::all(255), thickness, 8);
}





int main()
{



//	InitResnet18();

	//Classify(imread("/home/jurie/Pictures/resnet_test/1646501484_1_6.bmp"));

	static int iIndex=0;
	static int iFrameIndex=0;
	//DestroyResnet18();


	StartMonitor();

	if (1)
	{

#ifndef PREVIEW_OUTPUTRENDER

		initCameraUDPData();
		initOpenCVWindows();
		//InitVizSocket();
#endif

		cuda::GpuMat RGB_Output_Cuda;
		RGB_Output_Cuda.create(1080, 1920, CV_8UC3);
		RGB_Output_Cuda.step = 5760;

		cuda::GpuMat RGB_FrameInfo_Cuda;
		RGB_FrameInfo_Cuda.create(1024, 1024, CV_8UC3);
		RGB_FrameInfo_Cuda.step = 1024 * 3;

		cuda::GpuMat RGB_FrameInfo_Cuda_FullUpdate;
		RGB_FrameInfo_Cuda_FullUpdate.create(1024, 1024, CV_8UC3);
		RGB_FrameInfo_Cuda_FullUpdate.step = 1024 * 3;


		myThreadData.RGB_Output_Cuda = &RGB_Output_Cuda;
		Mat		MASK_L(1080, 1920, CV_8UC1, Scalar(0));
		myThreadData.MouseData1 = &MouseData1;
	//	imshow("RGB Output", RGB_Output_Cuda);


		pthread_t threads;
		int rc;
		rc = pthread_create(&threads, NULL, OutputRenderthread, (void *) &myThreadData);

		if (rc)
		{
			cout << "Error:unable to create thread," << rc << endl;
			exit(-1);
		}



		initPosUDPData();
		Mat RGB__Draw;
		Mat RGB_Output;
		Mat RGB_saving;
	//	std::vector<Yolo_Big::Detection> DetectionS;

		bool bstart = false;
		bool bCapture=false;

#ifdef PREVIEW_OUTPUTRENDER
		while (1)
		{
			std::this_thread::sleep_for(std::chrono::milliseconds(1000));
		};
#endif


		unsigned long UI_Frame_Counter=0;
		while (1)
		{
			std::this_thread::sleep_for(std::chrono::milliseconds(40));
			UI_Frame_Counter++;
//			if (GetAsyncKeyState('s'))
//			{
//				//if (GetAsyncKeyState(VK_LCONTROL))
//				{
//					std::ofstream outfile("settings.crm", std::ofstream::binary);
//					if (outfile.good())
//					{
//						outfile.seekp(ios_base::beg);
//						outfile.write((char *)&FourSettings, sizeof(WindowSettings) * 3);
//						outfile.close();
//						printf("settings saved to settings.crm\n");
//
//					}
//					else
//					{
//						printf("could not load settings\n");
//					}
//				}
//			}
//
//
//			if (GetAsyncKeyState('l'))
//			{
//				//if (GetAsyncKeyState(VK_LCONTROL))
//				{
//					std::ifstream  infile("settings.crm", std::ofstream::binary);
//					if (infile.good())
//					{
//						infile.seekg(ios_base::beg);
//						infile.read((char *)&FourSettings, sizeof(WindowSettings) * 3);
//						infile.close();
//						printf("settings loaded to settings.crm\n");
//					}	else
//					{
//						printf("could not save settings\n");
//					}
//				}
//
//				UpdateSettingsWindow();
//			//	while (GetAsyncKeyState('L'));
//			}


			if (GetAsyncKeyState('g'))
			{
				cudaLookUpFullKey();
		//		while (GetAsyncKeyState('F'));
			}


			if (GetAsyncKeyState('r'))//||(UI_Frame_Counter%10)==0
			{
				cudaLookUpReset(0);
			//	while (GetAsyncKeyState('R'));
			}

			if (GetAsyncKeyState('f'))//||(UI_Frame_Counter%10)==0
			{
				cudaLookUpReset(1);
			//	while (GetAsyncKeyState('R'));
			}

//			if (GetAsyncKeyState('1'))
//			{
//				iUpdateIndex = 0;
//				UpdateSettingsWindow();
//				SetOnAirLookup(0);
//				while (GetAsyncKeyState('1'));
//
//				setWindowTitle("Settings","Settings 1");
//				setWindowTitle("RGB Output", "Settings 1");
//			}
//			if (GetAsyncKeyState('2'))
//			{
//				iUpdateIndex = 1;
//				UpdateSettingsWindow();
//				SetOnAirLookup(1);
//				while (GetAsyncKeyState('2'));
//
//				setWindowTitle("Settings", "Settings 2");
//				setWindowTitle("RGB Output", "Settings 2");
//			}
//			if (GetAsyncKeyState('3'))
//			{
//				iUpdateIndex = 2;
//				UpdateSettingsWindow();
//				SetOnAirLookup(2);
//				while (GetAsyncKeyState('3'));
//
//				setWindowTitle("Settings", "Settings 3");
//				setWindowTitle("RGB Output", "Settings 3");
//			}
			if (GetAsyncKeyState('P'))
			{
				//InKey->ClearAll();
				//InFill->ClearAll();
			//	while (GetAsyncKeyState('P'));
			}


			if (GetAsyncKeyState('o'))
			{
				bClearOutPut=true;
				//InPut->ClearOutPut();
			//	while (GetAsyncKeyState('O'));
			}


			if (GetAsyncKeyState('b'))
			{
				bBypass = !bBypass;
			//	while (GetAsyncKeyState('B'));
			}

			if (GetAsyncKeyState('h'))
			{
				for(int x=0;x<10;x++)
					printf("\n\r");
				//printf("'1','2','3' � select user updateble lookup tables\n\r");
				printf("'q' chroma snapshot update \n\r");
				printf("'a' PaintItBack snapshot update \n\r");
				printf("'r' reset chroma lookup table\n\r");
				printf("'f' reset PaintItBack lookup table\n\r");
				printf("'o' to clear output buffer frame list\n\r");
				printf("'i' display frame info\n\r");
				printf("ctl+'l' load settings\n\r");
				printf("ctl+'s' save settings\n\r");

				for (int x = 0; x < 10; x++)
					printf("\n\r");
			//    while (GetAsyncKeyState('H'));
			}



//			if (GetAsyncKeyState('A'))
//			{
//				bSmall = true;
//			}else
//				bSmall = false;
			static unsigned int bFrameTimer = 0;
			bFrameTimer++;


			if (GetAsyncKeyState('w'))
			{

				bTakeOutput = 0;
				bAutoTrain=true;

			}else
			{
				bAutoTrain=false;
			}

			if (GetAsyncKeyState('z'))
			{
				bTrackReset=true;
			}
			if (GetAsyncKeyState('c'))
			{
				bCapture=true;
			}else
			{
				bCapture=false;
			}

			if (GetAsyncKeyState('q'))
			{
				SetOnAirLookup(0);
				iUpdateIndex = 0;
				UpdateSettingsWindow();

				iLastCheck = 0;
				bTakeOutput = -1;
				bDoPaintBack=false;
			//	if (!GetAsyncKeyState(VK_LCONTROL))
			//		bSafeSnapshot = true;
				mtxScreenCard.lock();
				bTakeMask = true;
			//	std::cout << "Snap Shot" << std::endl;

				mtxScreenCard.unlock();
				setWindowTitle("RGB Output", "Chroma");
				setWindowTitle("Settings","Settings Chroma");
			//	while (GetAsyncKeyState('Q'));
			}


			if (GetAsyncKeyState('a'))
			{

				SetOnAirLookup(1);
				iUpdateIndex = 1;
				UpdateSettingsWindow();
				iLastCheck = 0;
				bTakeOutput = -1;
				bDoPaintBack=true;
			//	if (!GetAsyncKeyState(VK_LCONTROL))
			//		bSafeSnapshot = true;
				mtxScreenCard.lock();
				bTakeMask = true;
				//std::cout << "Snap Shot" << std::endl;
				mtxScreenCard.unlock();
				setWindowTitle("RGB Output", "PaintItBack");
				setWindowTitle("Settings","PaintItBack");
			//	while (GetAsyncKeyState('Q'));
			}


			if (GetAsyncKeyState(VK_F1))
			{
				bTakeOutput = 0;
			//	while (GetAsyncKeyState(VK_F1));
			}

			if (GetAsyncKeyState(VK_F2))
			{
				bTakeOutput = 1;
			//	while (GetAsyncKeyState(VK_F2));
			}

			if (GetAsyncKeyState(VK_F3))
			{
				bTakeOutput = 2;
		//		while (GetAsyncKeyState(VK_F3));
			}
			if (GetAsyncKeyState(VK_F4))
			{
				bTakeOutput = 3;
				//		while (GetAsyncKeyState(VK_F3));
			}
			if (GetAsyncKeyState(VK_F5))
			{
				bTakeOutput = 4;
				//		while (GetAsyncKeyState(VK_F3));
			}

			if (GetAsyncKeyState(VK_F10))
				CameraZero();

			if (GetAsyncKeyState('s'))
			{
				std::string FileAndPathName;
				std::time_t result = std::time(nullptr);
				std::string  FileName= toString(result);
				FileAndPathName="/home/jurie/Pictures/yolov5_soccer_training/"+FileName;
				std::thread t1(SaveImageThread,RGB_saving.clone(),iIndex++,FileAndPathName);
				t1.join();
				if(iIndex==4)
					iIndex=0;
			}
//				iLastCheck = 1;
//				if (!GetAsyncKeyState(VK_LCONTROL))
//					bSafeSnapshot = true;
//
//			//	while (GetAsyncKeyState('W'));
//			}
//		//	Rect2d r = selectROI(im);

			if (myThreadData.bUpdateRGB_Preview)
			{
				myThreadData.bUpdateRGB_Preview = false;
			//	std::cout << "waiting" << std::endl;
				mtxScreenCard.lock();


				RGB_Output_Cuda.download(RGB_Output);
				RGB_saving=RGB_Output.clone();
				iFrameIndex++;
			//	std::cout << iFrameIndex << std::endl;
				if(0)
				if(iFrameIndex==50)
				{
					iFrameIndex=0;
					std::string FileAndPathName;
					std::time_t result = std::time(nullptr);
					std::string  FileName= toString(result);
					FileAndPathName="/home/jurie/Pictures/yolov5_soccer_training/"+FileName;
					std::thread t1(SaveImageThread,RGB_Output.clone(),iIndex++,FileAndPathName);
					t1.join();
					if(iIndex==4)
						iIndex=0;
				}
				DrawSnapShotDetections_clean(&RGB_Output,bTrackReset);
				//imwrite("/home/jurie/Documents/textures/from_camera.bmp",RGB_Output);
				//RGB_Output.cols=1920*2;
				//RGB_Output.rows=1080/2;

			//	DrawSnapShotDetectionsPTR(&RGB_Output,bTrackReset);
				if(bCapture)
					writeframe(RGB_Output.clone());
				bTrackReset=false;
				DrawCameraData(&RGB_Output);
				//RGB_Output.cols=1920;
				//RGB_Output.rows=1080;
				//writeframe(RGB_Output);
				bstart = true;
			//	bTakeOutput = 0;

			//	bTakeMask = true;
				mtxScreenCard.unlock();
			}
			if (bstart)
			{
				RGB__Draw = RGB_Output.clone();
				cv::Rect myROI(MouseData1.iXUpDynamic, MouseData1.iYUpDynamic, MouseData1.iXDownDynamic - MouseData1.iXUpDynamic, MouseData1.iYDownDynamic - MouseData1.iYUpDynamic);
				if((0 <= myROI.x && 0 <= myROI.width && myROI.x + myROI.width <= RGB__Draw.cols &&
					0 <= myROI.y && 0 <= myROI.height && myROI.y + myROI.height <= RGB__Draw.rows))
				{
					MouseMutex.lock();
					Mat RGB__Draw_Small = RGB__Draw(myROI);
					Mat RGB__Draw_SmallEnlarge;
					Size ssize = RGB__Draw_Small.size();
					if (!ssize.empty())
					{
						cv::resize(RGB__Draw_Small, RGB__Draw_SmallEnlarge, Size((MouseData1.iXDownDynamic - MouseData1.iXUpDynamic) * 25, (MouseData1.iYDownDynamic - MouseData1.iYUpDynamic) * 25), 0, 0, INTER_NEAREST);
						RGB__Draw_SmallEnlarge.copyTo(RGB__Draw.rowRange(0, RGB__Draw_SmallEnlarge.rows).colRange(0, RGB__Draw_SmallEnlarge.cols));
					}
					MouseMutex.unlock();
					bEnableClick = true;
				}
				else
				{
					bEnableClick = false;
				}

				rectangle(RGB__Draw, Point(MouseData1.iXUpDynamic, MouseData1.iYUpDynamic), Point(MouseData1.iXDownDynamic, MouseData1.iYDownDynamic), Scalar(255, 255, 255), 1, 8, 0);
				circle(RGB__Draw,Point(MouseData1.x,MouseData1.y),20,Scalar(255,255,255),3);
			//	RGB__Draw.cols=1920*2;
			//	RGB__Draw.rows=1080/2;


				DrawMouseText(&RGB__Draw,"Hello World",cv::Point(50,50));
				DrawOutputThreadData(&RGB__Draw);
				imshow("RGB Output", RGB__Draw);
				RGB__Draw.release();
			}

			if(0)
			if ((bFrameTimer%20)==0)
			{
				Launch_Frame_Info(&RGB_FrameInfo_Cuda);
				imshow("Frame Info", RGB_FrameInfo_Cuda);
			}

			if (GetAsyncKeyState('i'))
			{
				Launch_Frame_Info(&RGB_FrameInfo_Cuda);
				imshow("Frame Info", RGB_FrameInfo_Cuda);
			//	while (GetAsyncKeyState('I'));
			}

			UpdateLookupFromMouse();
			UpdateKeyState();


//			if (iKey == 49)
//			{
//				bTakeOutput = 1;
//
//
//			}
			if (GetAsyncKeyState(27))//"Esc"
			{
				// Do some clean up and free memory, c++ garbage collector doesn't clean up some things.
//				delete RGB__Draw.data;
				bExitWorkerThread = true;
				std::this_thread::sleep_for(std::chrono::milliseconds(100));
				EndLoop();
				std::this_thread::sleep_for(std::chrono::milliseconds(100));
				std::cout << "Exit" << std::endl;
				break;
//				bExite = true;
//				printf("Press 'Y' to exit or 'n' to cancel\n\r");
//				iExitCount = 0;
			}
			if (bExite)
			{
				iExitCount++;
			//	printf("%d\n\r", iExitCount);
				if (iExitCount == 100)
				{
					bExite = false;
					printf("Exit process canceled Exit process canceledExit process canceledExit process canceled\n\r");
				}
			}

			if (GetAsyncKeyState('y'))
				if (bExite)
				{
					//ExitMonitor();
					bExitWorkerThread = true;

					std::this_thread::sleep_for(std::chrono::milliseconds(1000));
					break;

				}

			if (GetAsyncKeyState('n'))
				//if (bExite)
					bExite = false;
		};


	}/*else
	ReadFileFast(true);*/
//	hipProfilerStop();
	hipError_t cudaStatus;
	 ExitMonitor();

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	std::cout << "End Cuda" << std::endl;
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}


	return 0;
}

// Helper function for using CUDA to add vectors in parallel.




